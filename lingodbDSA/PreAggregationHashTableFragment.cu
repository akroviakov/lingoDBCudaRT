#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "GrowingBuffer.cuh"
#include "LazyJoinHashtable.cuh"
#include "PreAggregationHashtable.cuh"
#include "PrefixSum.cuh"
#include "lock.cuh"
#include "util.h"

size_t getSharedMemorySize() {
    int device;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);
    return static_cast<size_t>(deviceProp.sharedMemPerBlock);
}

size_t getNumberOfElementsInSMEM(size_t elementSize) {
    return getSharedMemorySize()/elementSize;
}

std::pair<size_t, size_t> getHtSizeMask(size_t numElements, size_t elementSize){
    size_t size = max(PreAggregationHashtable::nextPow2(numElements * 1.25), 1ull);
    return {size*elementSize, size-1};
}

constexpr size_t KiB = 1024;
constexpr size_t MiB = 1024 * KiB;
constexpr size_t GiB = 1024 * MiB;
constexpr size_t heapSize = 3 * GiB;

constexpr int initialCapacity = INITIAL_CAPACITY;
constexpr float selectivity = 0.8;
constexpr int numbersThreshold = 10;
constexpr int LTPredicate = (int)numbersThreshold * selectivity;
__device__ volatile int globalLock = 0;


__device__ uint32_t hashInt32(int32_t key) {
    key ^= key >> 16;
    key *= 0x85ebca6b;
    key ^= key >> 13;
    key *= 0xc2b2ae35;
    key ^= key >> 16;
    return key;
}

struct HashIndexedViewEntry {
    HashIndexedViewEntry* next;
    uint64_t hashValue;
    bool nullFlag;
    int32_t key;
    int32_t value;
};

constexpr int typeSize = sizeof(HashIndexedViewEntry);
constexpr int warp_size = 32;

enum class KernelType{
    Naive = 0,
    WarpLevel = 1,
    WarpLevelPickAnyFree = 2
};

enum class HashIndexedViewBuilderType{
    BufferToSM = 1,
    BufferToGPU = 2
};

__global__ void growingBufferFillTB(int** input, int numPredColumns, int size, GrowingBuffer* finalBuffer) {
    const int warp_count = (blockDim.x + (warp_size-1)) / warp_size;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    const int lane = threadIdx.x % warp_size;
    const int warpId = threadIdx.x / warp_size;

    extern __shared__ char sharedMem[];
    // Ensure correct memory alignment
    GrowingBuffer* myBuf = reinterpret_cast<GrowingBuffer*>(sharedMem);
    uint64_t* counter = reinterpret_cast<uint64_t*>(sharedMem + sizeof(GrowingBuffer));
    HashIndexedViewEntry** writeCursor = reinterpret_cast<HashIndexedViewEntry**>(sharedMem + sizeof(GrowingBuffer) + sizeof(uint64_t));

    if (threadIdx.x == 0) {
        new (myBuf) GrowingBuffer(initialCapacity, typeSize);
        *counter = 0;  // Initialize the counter
    }
    __syncthreads();
    uint64_t myIdx = 0;

    int roundedSize = ((size + 31) / 32) * 32;
    for (int i = globalTid; i < roundedSize; i += numThreadsTotal) {
        bool pred = (i < size);
        int colIdx = 0;
        // while (pred && colIdx < numPredColumns) {
            pred &= (input[colIdx][i] < LTPredicate);
            // colIdx++;
        // }
        const int maskWriters = __ballot_sync(0xFFFFFFFF, pred);
        if(!lane){
            myIdx = atomicAdd((unsigned long long*)counter, (unsigned long long)__popc(maskWriters));
        }
        myIdx = __shfl_sync(maskWriters, myIdx, 0);
        __syncthreads();

        if (threadIdx.x == 0) {
            *writeCursor = (HashIndexedViewEntry*)myBuf->getValues().prepareWriteFor(*counter);
            *counter = 0;
        }
        __syncthreads();
        if (pred) {
            HashIndexedViewEntry* writeTo= *writeCursor;
            writeTo[myIdx].key = input[0][i];
            writeTo[myIdx].hashValue = hashInt32(writeTo[myIdx].key);
            writeTo[myIdx].value = input[0][i];
            writeTo[myIdx].nullFlag = false;
        }
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        acquire_lock(&globalLock);
        finalBuffer->getValues().merge(myBuf->getValues());
        __threadfence();
        release_lock(&globalLock);
    }
}
__global__ void growingBufferInit(GrowingBuffer* finalBuffer) {
    if(blockDim.x * blockIdx.x + threadIdx.x == 0){
        new(finalBuffer) GrowingBuffer(initialCapacity, typeSize, false);
    }
}
__global__ void growingBufferFill(int** input, int numPredColumns, int size, GrowingBuffer* finalBuffer) {
    const int warp_count = (blockDim.x + (warp_size-1)) / warp_size;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    const int lane = threadIdx.x % warp_size;
    const int warpId = threadIdx.x / warp_size;
    extern __shared__ char sharedMem[];
    GrowingBuffer* warpBuffers = reinterpret_cast<GrowingBuffer*>(sharedMem);
    GrowingBuffer* currentWarpBuffer = &warpBuffers[warpId];
    if(lane == 0){
        new (currentWarpBuffer) GrowingBuffer(initialCapacity, typeSize);
    }
    __syncwarp();
    int roundedSize = ((size + 31) / 32) * 32; 
    for (int i = globalTid; i < roundedSize; i += numThreadsTotal) {
        bool pred{i < size};
        int colIdx=0;
        // while(pred && colIdx < numPredColumns){
            pred &= (input[colIdx][i] < LTPredicate);
            // colIdx++;
        // }
        // TODO: revisit (see FlexibleBuffer::insertWarpLevel())
        // if (pred) {
        //     HashIndexedViewEntry* writeCursor = (HashIndexedViewEntry*) currentWarpBuffer->getValues().insertWarpLevel();
        //     writeCursor->key = input[0][i];
        //     writeCursor->hashValue = hashInt32(writeCursor->key);
        //     writeCursor->value = input[0][i];
        //     writeCursor->nullFlag = false;
        // }

        // DO NOT USE __activemask(), threads from the same warp can arrive here at different time 
        //  and __activemask() would NOT block the entire warp, then in prepareWriteFor, the warp threads can again converge, but
        //  there would be already 2 leaders as one warp had two different masks due to __activemask() which is race condition!
        const unsigned int mask = __ballot_sync(0xFFFFFFFF, pred); 
        if(mask){
            HashIndexedViewEntry* writeCursor;
            const int leader{__ffs(mask)-1};
            if (lane == leader) {
                writeCursor = (HashIndexedViewEntry*) currentWarpBuffer->getValues().prepareWriteFor(__popc(mask));
            }
            if (pred) {
                const int threadOffset = __popc(mask & ((1U << lane) - 1));
                writeCursor = (HashIndexedViewEntry*) __shfl_sync(mask, (uintptr_t)writeCursor, leader);
                writeCursor[threadOffset].key = input[0][i];
                writeCursor[threadOffset].hashValue = hashInt32(writeCursor[threadOffset].key);
                // printf("KEY: %d, HASH: %llu\n", input[0][i], writeCursor[threadOffset].hashValue);
                writeCursor[threadOffset].value = input[0][i];
                writeCursor[threadOffset].nullFlag = false;
            }
        }
    }
    __syncthreads();
    for(int wid = 1; wid < warp_count; wid++){
        if(warpId == wid && lane == 0){
            warpBuffers[0].getValues().merge(warpBuffers[warpId].getValues());
        }
        __syncthreads();
    }
    __syncthreads();
    if(threadIdx.x == 0){
        acquire_lock(&globalLock);
        finalBuffer->getValues().merge(warpBuffers[0].getValues());
        __threadfence();
        release_lock(&globalLock);
    }
}

__global__ void freeKernel(GrowingBuffer* finalBuffer, HashIndexedView* view) {
    finalBuffer->~GrowingBuffer();
    // view->~HashIndexedView();
}

__global__ void printHashIndexedView(HashIndexedView* view) {
    view->print();
}

__device__ void printEntry(uint8_t* entryPtr){
    HashIndexedViewEntry* structPtr = reinterpret_cast<HashIndexedViewEntry*>(entryPtr);
    printf("{key=%d,val=%d,hash=%llu,next=%p},", structPtr->key, structPtr->value, structPtr->hashValue, structPtr->next);
}

__global__ void printPreAggregationHashtable(PreAggregationHashtable* ht, bool printEmpty=false) {
    printf("---------------------PreAggregationHashtable [%p]-------------------------\n", ht);
    for(int p = 0; p < 64; p++){
        for(int i = 0; i < ht->ht[p].hashMask+1; i++){
            HashIndexedViewEntry* curr = reinterpret_cast<HashIndexedViewEntry*>(ht->ht[p].ht[i]);
            if(!printEmpty && !curr){continue;}
            printf("[PARTITION %d, htEntryIdx=%d]", p, i);
            while(curr){
                printf(", {ptr=%p, next=%p, KEY: %d, AGG: %d}", curr, curr->next, curr->key, curr->value);
                curr = curr->next;
            }
            printf("\n");
        }
    }
    printf("------------------[END] PreAggregationHashtable [%p]----------------------\n", ht);

}

struct HIVEntry{
    HashIndexedView::Entry* head{nullptr};
    HashIndexedView::Entry* tail{nullptr};
    int64_t writeOutPos{-1};
    // size_t chainLength{0};
};

__device__ void atomicAppendSubList(HashIndexedView::Entry** globalHt, const size_t pos, 
        HashIndexedView::Entry* subListHead, 
        HashIndexedView::Entry* subListTail) {
    HashIndexedView::Entry* currentHead = globalHt[pos];
    HashIndexedView::Entry* old = currentHead;
    do {
        currentHead = old;
        if(subListHead != subListTail){ // We set tail to head on the first write to head, if they do not match -> sub-list length > 1, need to adjust tail
            subListTail->next = currentHead; 
        } else // if subListHead is not different from tail -> sub-list length 1, only adjust head
            subListHead->next = currentHead; 
        subListHead = tag(subListHead, currentHead, subListHead->hashValue);
        old = (HashIndexedView::Entry*) atomicCAS((unsigned long long*)&globalHt[pos], (unsigned long long)currentHead, (unsigned long long)subListHead);
    } while (old != currentHead);
}

__device__ __forceinline__ void atomicAppendSMEM(HIVEntry* globalHt, const size_t pos, HashIndexedView::Entry* newNode) {
    HashIndexedView::Entry* currentHead = globalHt[pos].head;
    HashIndexedView::Entry* old = currentHead;
    do {
        currentHead = old;
        newNode->next = currentHead; 
        newNode = tag(newNode, currentHead, newNode->hashValue);
        old = (HashIndexedView::Entry*) atomicCAS((unsigned long long*)&globalHt[pos].head, (unsigned long long)currentHead, (unsigned long long)newNode);
    } while (old != currentHead);
    if(!old) { // We executed the first write (meaning head was nullptr before): set tail to head. No other write to this head would update it anymore -> thread safe.
        globalHt[pos].tail = newNode;
    }
}

__device__ __forceinline__ void atomicAppend(HashIndexedView::Entry** globalHt, const size_t pos, HashIndexedView::Entry* newNode) {
    HashIndexedView::Entry* currentHead = globalHt[pos];
    HashIndexedView::Entry* old = currentHead;
    do {
        currentHead = old;
        newNode->next = currentHead; 
        newNode = tag(newNode, currentHead, newNode->hashValue);
        old = (HashIndexedView::Entry*) atomicCAS((unsigned long long*)&globalHt[pos], (unsigned long long)currentHead, (unsigned long long)newNode);
    } while (old != currentHead);
}


template<HashIndexedViewBuilderType Qimpl = HashIndexedViewBuilderType::BufferToSM>
__global__ void buildHashIndexedViewAdvancedSMEMWarpLevel(GrowingBuffer* buffer, HashIndexedView* view) {
    const int warpCount = (blockDim.x + (warp_size-1)) / warp_size;
    const int warpId = threadIdx.x / warp_size;
    const int warpLane = threadIdx.x % warp_size;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    // Num warps - 2^5. 
    // 2^11 / 2^5 = 2^6; - 64 slots per warp, not bad.
    const int numWarps = blockDim.x / warp_size; // 2^1 ... 2^5
    const int powerTwoTemp{11};
    const int subHtSize{1 << powerTwoTemp};
    const int warpSubHtSize{subHtSize / numWarps};
    // if(!threadIdx.x){
    //     printf("numWarps=%d, warpSubHtSize=%d\n", numWarps, warpSubHtSize);
    // }
    // Example:
    // 1024 threads means 32 warps.
    // We can host 2^11 slots for 2^5 warps, each warp gets 2^6 slots.

    const size_t scracthPadMask{subHtSize - 1};
    __shared__ HIVEntry scracthPad[subHtSize];
    HIVEntry* myScratchPadStart = &scracthPad[warpId * warpSubHtSize];
    const size_t myScracthPadMask{warpSubHtSize - 1};

    for(int i=warpLane; i < warpSubHtSize;i+=blockDim.x){
        myScratchPadStart[i].writeOutPos = -1ll;
        myScratchPadStart[i].head = nullptr;
        myScratchPadStart[i].tail = nullptr;
    }
    
    __syncwarp();

    auto& values = buffer->getValues();
    int bufferIdxStart{0};
    int bufferIdxStep{0};
    int bufferEntryIdxStart{0};
    int bufferEntryIdxStep{0};
    if constexpr(Qimpl == HashIndexedViewBuilderType::BufferToSM){
        bufferIdxStart = blockIdx.x;
        bufferIdxStep = gridDim.x;
        bufferEntryIdxStart = threadIdx.x;
        bufferEntryIdxStep = blockDim.x;
    }
    else{
        bufferIdxStart = 0;
        bufferIdxStep = 1;
        bufferEntryIdxStart = globalTid;
        bufferEntryIdxStep = numThreadsTotal;
    }
    // int conflictCnt{0};
    for(int bufIdx=bufferIdxStart; bufIdx<values.buffers.count; bufIdx+=bufferIdxStep){ // Buffer-per-block
        auto* buffer = &values.buffers.payLoad[bufIdx];
        for (int bufEntryIdx = bufferEntryIdxStart; bufEntryIdx < buffer->numElements; bufEntryIdx+=bufferEntryIdxStep) { // Entry-per-warp
            HashIndexedView::Entry* entry = (HashIndexedView::Entry*) &buffer->ptr[bufEntryIdx * typeSize];
            size_t hash = (size_t) entry->hashValue;
            const size_t posGlobal = hash & view->htMask;
            const size_t posLocal = hash & myScracthPadMask;

            HashIndexedView::Entry* localEntryCurrent = myScratchPadStart[posLocal].head;
            HashIndexedView::Entry* old = localEntryCurrent;

            const int64_t writeOutPos = atomicCAS((unsigned long long*)&myScratchPadStart[posLocal].writeOutPos, (unsigned long long)-1, (unsigned long long)posGlobal);
            // printf("[Considering] writeOutPos=%lld, posGlobal=%lu, posLocal=%lu, hash=%d\n", writeOutPos, posGlobal, posLocal, entry->hashValue);

            if(writeOutPos == -1 || writeOutPos == posGlobal){ // Was an empty SMEM slot, we just occupied it with posGlobal. OR matched the writeout position.
                atomicAppendSMEM(myScratchPadStart, posLocal, entry);
                // printf("[CACHED APPEND] writeOutPos=%lld, posLocal=%lu, hash=%d\n", writeOutPos, posLocal, entry->hashValue);

            } else { // If we have a collision (scratch pad's entry writeout position != entry's writeOut pos) -> write directly to global.
                atomicAppend(view->ht, posGlobal, entry);
                // conflictCnt++;
            }
        }
    }

    __syncwarp();
    for(int i=warpLane; i < warpSubHtSize;i+=blockDim.x){
        if(myScratchPadStart[i].head){
            atomicAppendSubList(view->ht, myScratchPadStart[i].writeOutPos, myScratchPadStart[i].head, myScratchPadStart[i].tail);
            // printf("[FINAL] SUBCHAIN LENGTH %d\n", scracthPad[i].chainLength);
        }
    }

    // __syncthreads();
    // if(!threadIdx.x){
    //     printf("[SMEM Warp Level] APPENDING CONFLICT %d\n", conflictCnt);
    // }
}

template<HashIndexedViewBuilderType Qimpl = HashIndexedViewBuilderType::BufferToSM>
__global__ void buildHashIndexedViewAdvancedSMEM(GrowingBuffer* buffer, HashIndexedView* view) {
    const int warpCount = (blockDim.x + (warp_size-1)) / warp_size;
    const int warpId = threadIdx.x / warp_size;
    const int warpLane = threadIdx.x % warp_size;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;

    const int powerTwoTemp{10};
    const size_t scracthPadMask{(1 << powerTwoTemp) - 1};
    __shared__ HIVEntry scracthPad[1 << powerTwoTemp];

    for(int i = threadIdx.x; i < scracthPadMask+1; i+=blockDim.x){
        scracthPad[i].writeOutPos = -1ll;
        scracthPad[i].head = nullptr;
        scracthPad[i].tail = nullptr;
    }
    __syncthreads();

    auto& values = buffer->getValues();
    int bufferIdxStart{0};
    int bufferIdxStep{0};
    int bufferEntryIdxStart{0};
    int bufferEntryIdxStep{0};
    if constexpr(Qimpl == HashIndexedViewBuilderType::BufferToSM){
        bufferIdxStart = blockIdx.x;
        bufferIdxStep = gridDim.x;
        bufferEntryIdxStart = threadIdx.x;
        bufferEntryIdxStep = blockDim.x;
    }
    else{
        bufferIdxStart = 0;
        bufferIdxStep = 1;
        bufferEntryIdxStart = globalTid;
        bufferEntryIdxStep = numThreadsTotal;
    }
    int conflictCnt{0};
    for(int bufIdx=bufferIdxStart; bufIdx<values.buffers.count; bufIdx+=bufferIdxStep){ // Buffer-per-block
        auto* buffer = &values.buffers.payLoad[bufIdx];
        for (int bufEntryIdx = bufferEntryIdxStart; bufEntryIdx < buffer->numElements; bufEntryIdx+=bufferEntryIdxStep) { // Entry-per-warp
            HashIndexedView::Entry* entry = (HashIndexedView::Entry*) &buffer->ptr[bufEntryIdx * typeSize];
            size_t hash = (size_t) entry->hashValue;
            const size_t posGlobal = hash & view->htMask;
            const size_t posLocal = hash & scracthPadMask;

            HashIndexedView::Entry* localEntryCurrent = scracthPad[posLocal].head;
            HashIndexedView::Entry* old = localEntryCurrent;

            const int64_t writeOutPos = atomicCAS((unsigned long long*)&scracthPad[posLocal].writeOutPos, (unsigned long long)-1, (unsigned long long)posGlobal);
            // printf("[Considering] writeOutPos=%lld, posGlobal=%lu, posLocal=%lu, hash=%d\n", writeOutPos, posGlobal, posLocal, entry->hashValue);

            if(writeOutPos == -1 || writeOutPos == posGlobal){ // Was an empty SMEM slot, we just occupied it with posGlobal. OR matched the writeout position.
                atomicAppendSMEM(scracthPad, posLocal, entry);
                // printf("[CACHED APPEND] writeOutPos=%lld, posLocal=%lu, hash=%d\n", writeOutPos, posLocal, entry->hashValue);

            } else { // If we have a collision (scratch pad's entry writeout position != entry's writeOut pos) -> write directly to global.
                atomicAppend(view->ht, posGlobal, entry);
                conflictCnt++;
            }
        }
    }

    __syncthreads();
    // if(!threadIdx.x){
    //     printf("[SMEM] APPENDING CONFLICT %d\n", conflictCnt);
    // }
    for(int i = threadIdx.x; i < scracthPadMask+1; i+=blockDim.x){
        if(scracthPad[i].head){
            atomicAppendSubList(view->ht, scracthPad[i].writeOutPos, scracthPad[i].head, scracthPad[i].tail);
            // printf("[FINAL] SUBCHAIN LENGTH %d\n", scracthPad[i].chainLength);
        }
    }
    
}

template<HashIndexedViewBuilderType Qimpl = HashIndexedViewBuilderType::BufferToSM>
__global__ void buildHashIndexedViewAdvanced(GrowingBuffer* buffer, HashIndexedView* view) {
    const int warpCount = (blockDim.x + (warp_size-1)) / warp_size;
    const int warpId = threadIdx.x / warp_size;
    const int warpLane = threadIdx.x % warp_size;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;

    const int powerTwoTemp{8};
    const size_t scracthPadMask{(1 << powerTwoTemp) - 1};
    HIVEntry scracthPad[1 << powerTwoTemp];
    auto& values = buffer->getValues();
    int bufferIdxStart{0};
    int bufferIdxStep{0};
    int bufferEntryIdxStart{0};
    int bufferEntryIdxStep{0};
    if constexpr(Qimpl == HashIndexedViewBuilderType::BufferToSM){
        bufferIdxStart = blockIdx.x;
        bufferIdxStep = gridDim.x;
        bufferEntryIdxStart = threadIdx.x;
        bufferEntryIdxStep = blockDim.x;
    }
    else{
        bufferIdxStart = 0;
        bufferIdxStep = 1;
        bufferEntryIdxStart = globalTid;
        bufferEntryIdxStep = numThreadsTotal;
    }

    // Naive Loop body: threads read the hash value and try to "insert" into ht[hash & view->htMask] which is the head of linked list. 
    // PROBLEM: few groups -> severe overhead.
    // DETAIL: But HTSize can be huge, it is of filtered scan size(!).
    // IDEA: ESTIMATE whether we have < SMEM_SIZE/PTR_SIZE. This estimate can be used to determine whether we can fit into SMEM (i.e., we have few groups).
    //       few groups -> fit in SMEM, aggregate locally (warp-level) and insert sub-list -> avoid severe bottleneck.

    for(int bufIdx=bufferIdxStart; bufIdx<values.buffers.count; bufIdx+=bufferIdxStep){ // Buffer-per-block
        auto* buffer = &values.buffers.payLoad[bufIdx];
        for (int bufEntryIdx = bufferEntryIdxStart; bufEntryIdx < buffer->numElements; bufEntryIdx+=bufferEntryIdxStep) { // Entry-per-warp
            HashIndexedView::Entry* entry = (HashIndexedView::Entry*) &buffer->ptr[bufEntryIdx * typeSize];
            size_t hash = (size_t) entry->hashValue;
            const size_t posGlobal = hash & view->htMask;
            const size_t posLocal = hash & scracthPadMask;

            HashIndexedView::Entry* localEntryCurrent = scracthPad[posLocal].head;
            HashIndexedView::Entry* old = localEntryCurrent;

            if (!scracthPad[posLocal].head || scracthPad[posLocal].writeOutPos == posLocal) { // if empty slot, don't care about write out pos
                // printf("[TID %d][bufEntryIdx=%d] hash=%lu, scracthPad[posLocal].head=%p,  writeOutPos=%lu, local=%lu == global=%lu\n", globalTid, bufEntryIdx, hash, scracthPad[posLocal].head, scracthPad[posLocal].writeOutPos, posLocal, posGlobal);
                
                entry->next = scracthPad[posLocal].head; 

                if (!scracthPad[posLocal].head) { // Insert first slot: head of list becomes the entry; entry->next is nullptr.
                    scracthPad[posLocal].writeOutPos = posGlobal;
                    scracthPad[posLocal].head = entry;
                } else if(!scracthPad[posLocal].tail){ // Second insert: push down head to tail
                    scracthPad[posLocal].tail = scracthPad[posLocal].head;
                }
                scracthPad[posLocal].head = entry;
                scracthPad[posLocal].head = tag(entry, scracthPad[posLocal].head, hash);  
                // scracthPad[posLocal].chainLength++;
            } 
            else { // non-empty slot AND different write out position.
                atomicAppendSubList(view->ht, scracthPad[posLocal].writeOutPos, scracthPad[posLocal].head, scracthPad[posLocal].tail);
                entry->next = nullptr; // Sublist appended, reset the new sublist.
                // printf("[CONFLICT] SUBCHAIN LENGTH %d\n", scracthPad[posLocal].chainLength);
                scracthPad[posLocal].writeOutPos = posGlobal;
                scracthPad[posLocal].head = entry;
                scracthPad[posLocal].tail = nullptr;
                scracthPad[posLocal].head = tag(entry, scracthPad[posLocal].head, hash);  
                // scracthPad[posLocal].chainLength=1;
            }
        }
    }
    for(int i = 0; i < 1 << powerTwoTemp; i++){
        if(scracthPad[i].head){
            atomicAppendSubList(view->ht, scracthPad[i].writeOutPos, scracthPad[i].head, scracthPad[i].tail);
            // printf("[FINAL] SUBCHAIN LENGTH %d\n", scracthPad[i].chainLength);
        }
    }
}

template<HashIndexedViewBuilderType Qimpl = HashIndexedViewBuilderType::BufferToSM>
__global__ void buildHashIndexedView(GrowingBuffer* buffer, HashIndexedView* view) {
    const int warpCount = (blockDim.x + (warp_size-1)) / warp_size;
    const int warpId = threadIdx.x / warp_size;
    const int warpLane = threadIdx.x % warp_size;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;

    auto& values = buffer->getValues();
    int bufferIdxStart{0};
    int bufferIdxStep{0};
    int bufferEntryIdxStart{0};
    int bufferEntryIdxStep{0};
    if constexpr(Qimpl == HashIndexedViewBuilderType::BufferToSM){
        bufferIdxStart = blockIdx.x;
        bufferIdxStep = gridDim.x;
        bufferEntryIdxStart = threadIdx.x;
        bufferEntryIdxStep = blockDim.x;
    }
    else{
        bufferIdxStart = 0;
        bufferIdxStep = 1;
        bufferEntryIdxStart = globalTid;
        bufferEntryIdxStep = numThreadsTotal;
    }


    // Naive Loop body: threads read the hash value and try to "insert" into ht[hash & view->htMask] which is the head of linked list. 
    // PROBLEM: few groups -> severe overhead.
    // DETAIL: But HTSize can be huge, it is of filtered scan size(!).
    // IDEA: ESTIMATE whether we have < SMEM_SIZE/PTR_SIZE. This estimate can be used to determine whether we can fit into SMEM (i.e., we have few groups).
    //       few groups -> fit in SMEM, aggregate locally (warp-level) and insert sub-list -> avoid severe bottleneck.
    for(int bufIdx=bufferIdxStart; bufIdx<values.buffers.count; bufIdx+=bufferIdxStep){ // Buffer-per-block
        auto* buffer = &values.buffers.payLoad[bufIdx];
        for (int bufEntryIdx = bufferEntryIdxStart; bufEntryIdx < buffer->numElements; bufEntryIdx+=bufferEntryIdxStep) { // Entry-per-warp
            HashIndexedView::Entry* entry = (HashIndexedView::Entry*) &buffer->ptr[bufEntryIdx * typeSize];
            size_t hash = (size_t) entry->hashValue;
            auto pos = hash & view->htMask;
            HashIndexedView::Entry* current = view->ht[pos];
            HashIndexedView::Entry* old = current;
            do {
                current = old;
                entry->next = current; 
                entry = tag(entry, current, hash);  
                old = (HashIndexedView::Entry*) atomicCAS((unsigned long long*)&view->ht[pos], (unsigned long long)current, (unsigned long long)entry);
            } while (old != current);
        }
    }
}

__device__ int collisionCnt{0};
__device__ int matchCnt{0};
__device__ int keyCnt{0};

__global__ void buildPreAggregationHashtableFragments(int** probeCols, int numProbeCols, int probeColsLength, HashIndexedView* view, FlexibleBuffer** globalOutputs, size_t* htSizes) {
    const int warpCount = (blockDim.x + (warp_size-1)) / warp_size;
    const int warpId = threadIdx.x / warp_size;
    const int warpLane = threadIdx.x % warp_size;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    const int globalWarpID = globalTid / warp_size;

    constexpr size_t outputMask = PreAggregationHashtableFragment::numOutputs - 1;
    constexpr size_t htMask = PreAggregationHashtableFragment::hashtableSize - 1;
    constexpr size_t htShift = 6; 
    
    /*
        Shared memory is mainly used to host ht scratchpad of multiple warp-level PreAggregationHashtableFragment.
    */
    extern __shared__ char sharedMem[];
    HashIndexedViewEntry d;
    PreAggregationHashtableFragment* preAggrHTFrags = reinterpret_cast<PreAggregationHashtableFragment*>(sharedMem);
    PreAggregationHashtableFragment* myPreAggrHTFrag = &preAggrHTFrags[warpId];
    if(!warpLane){
        new (myPreAggrHTFrag) PreAggregationHashtableFragment(typeSize);
    }
    __syncwarp();

    // printf("[%d/%d] myPreAggrHTFrag =  %p", globalTid, warpId, myPreAggrHTFrag);
 
    if(!globalTid){
        collisionCnt = 0;
        matchCnt = 0;
    }

    // iterate over probe cols
    int probeColIdxStart = globalTid;
    int probeColIdxStep = numThreadsTotal;
    int roundedSize = ((probeColsLength + 31) / 32) * 32; 
    for(int probeColIdx = probeColIdxStart; probeColIdx < roundedSize; probeColIdx+=probeColIdxStep){
        // const int maskLoop = __ballot_sync(0xFFFFFFFF, probeColIdx < probeColsLength);
        if(probeColIdx >= probeColsLength){break;}
        const int val = probeCols[0][probeColIdx];
        const uint32_t hash = hashInt32(val);
        const uint32_t pos = hash & view->htMask;
        HashIndexedViewEntry* current = reinterpret_cast<HashIndexedViewEntry*>(view->ht[pos]); // we have one view here (can have more in case of joins)
        bool foundMatch{false};
        while(current){ // probe HashIndexedView
            foundMatch = (current->hashValue == hash && current->key == val);
            if (foundMatch) {break;}
            current = current->next;
        }
        // const int maskFound = __ballot_sync(maskLoop, foundMatch);
        if(foundMatch){
            const int groupVal = current->value;
            const uint32_t hashGroupCol = hashInt32(groupVal); // pack group cols: %132 = util.pack %131, %65, then calculate hash: db.hash %132
            HashIndexedViewEntry* outputEntry = reinterpret_cast<HashIndexedViewEntry*>(myPreAggrHTFrag->ht[hashGroupCol >> htShift & htMask]);
            // Warp barrier, avoid race conditions (possible writes) when probing ht, outputEntry is read in a thread-safe manner.
            // __syncwarp(maskFound); // synchronize read for all threads that found a match
            // At this point, warp threads can reference the same outputEntry (or even have the same key), we have 2 scenarios in this case:
            //  1. No insert is needed (matching key) -> atomically aggregate (lock/unlock).
            //  2. Insert is needed -> each warp thread will insert an element (no intra-warp aggregation on key-matching ht insertion - TODO).

            // Entry is backed by the FlexibleBuffer, ht only offers a slot for a pointer to it and we read the pointer synchronously above.
            bool needInsert{false};
            if(!outputEntry){ 
                needInsert = true; // if no entry found (nullptr) at position
            } else {
                if(outputEntry->hashValue == hashGroupCol){ 
                    if(outputEntry->key == groupVal){
                        atomicAdd(&matchCnt, 1);
                        needInsert = false; // if found entry, hash and key match
                    } else { 
                        atomicAdd(&collisionCnt, 1);
                        needInsert = true; // if key doesn't match, collision -> insert
                    }
                } else { 
                    needInsert = true; // if hash doesn't match
                }
            }
            // __syncwarp(maskFound);
            // const int maskInsert = __ballot_sync(0, needInsert);
            if(needInsert){
                outputEntry = reinterpret_cast<HashIndexedViewEntry*>(myPreAggrHTFrag->insert(hashGroupCol));
                // printf("[%d]SETTING KEY %d of %p\n",threadIdx.x, groupVal,outputEntry);
                outputEntry->key = groupVal; // write key
                outputEntry->value = 0; // initialize value (aggregate)
                atomicExch((unsigned long long*)&myPreAggrHTFrag->ht[hash >> htShift & htMask], (unsigned long long)outputEntry);
            }
            PreAggregationHashtable::lock(reinterpret_cast<PreAggregationHashtable::Entry*>(outputEntry), 0);
            // atomicAdd(&outputEntry->value, probeCols[numProbeCols-1][probeColIdx]);
            outputEntry->value += probeCols[numProbeCols-1][probeColIdx]; // update aggregate with non-key column.
            PreAggregationHashtable::unlock(reinterpret_cast<PreAggregationHashtable::Entry*>(outputEntry), 0);

            // if(groupVal != outputEntry->value){
                // printf("[WarpId %d | TID %d] Key %d, Val %d\n", warpId, globalTid, groupVal, outputEntry->value);
            // }
        }
    }
    __syncthreads();

    if(!warpLane){
        int su = 0;
        for(int i = 0; i < 64; i++){
            if(myPreAggrHTFrag->outputs[i]){
                atomicAdd((unsigned long long*)&htSizes[i], (unsigned long long)myPreAggrHTFrag->outputs[i]->getLen());
                su += myPreAggrHTFrag->outputs[i]->getLen();
            }
        }
        memcpy(&globalOutputs[globalWarpID*64], myPreAggrHTFrag->outputs, sizeof(FlexibleBuffer *) * 64);
    }
    ///////////////////////////////////////////
    // if(!warpLane){
    //     printf("[%d | WarpId %d] preAggrHTFrag length: %lu\n", blockIdx.x, warpId, myPreAggrHTFrag->len);
    // }
    // if(!threadIdx.x){ // check that preAggrHT is initialized
    //     printf("Collisions %d, matches %d\n", collisionCnt, matchCnt);
    // }

    if(!warpLane){ // DEBUG PRINT (for singlewarp/singlethreaded use)
        // myPreAggrHTFrag->print(printEntry);
    }
}  


struct BuildStatsRadixCache{
    int scanSize{0};
    int size{0};
    int hits{0};
    int keyMismatch{0};
    int hashMismatch{0};
    int evicted{0};

    __device__ void print(){
        printf("[PreAggregationHTFrag BuildStats] : scanSize=%d, cacheSize=%d, cacheHits=%d, cacheKeyMismatch=%d, hashMismatch=%d, evicted=%d\n",scanSize, size, hits, keyMismatch, hashMismatch, evicted);
    }
};
__device__ BuildStatsRadixCache buildStats;
__global__ void buildPreAggregationHashtableFragmentsAdvanced(int** probeCols, 
        int numProbeCols, 
        int probeColsLength, 
        HashIndexedView* view, 
        FlexibleBuffer** globalOutputs, 
        size_t* htSizes) 
    {
    //  1024 threads per block, the maximum registers per thread is 64
    const int warpCount = (blockDim.x + (warp_size-1)) / warp_size;
    const int warpId = threadIdx.x / warp_size;
    const int warpLane = threadIdx.x % warp_size;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    const int globalWarpID = globalTid / warp_size;

    constexpr size_t outputMask = PreAggregationHashtableFragment::numOutputs - 1;
    constexpr size_t htShift = 6; 


    const int powerTwoTemp{10};
    const int scracthPadSize{1 << powerTwoTemp};
    const size_t scracthPadMask{scracthPadSize - 1};

    __shared__ PreAggregationHashtableFragmentSMEM::Entry* scracthPad[scracthPadSize];
    for(int i = threadIdx.x; i < scracthPadSize; i+=blockDim.x){
        scracthPad[i] = nullptr;
    }
    if(threadIdx.x == 0){
        buildStats = BuildStatsRadixCache{};
        buildStats.scanSize = probeColsLength;
        buildStats.size=scracthPadSize;
    }
    __syncthreads();


    if(!warpLane){
        // Warp stores its PreAggregationHashtableFragmentSMEM on heap, the pointer is cached on 
        globalOutputs[globalWarpID*64] = reinterpret_cast<FlexibleBuffer*>(memAlloc(sizeof(PreAggregationHashtableFragmentSMEM)));
        new(reinterpret_cast<PreAggregationHashtableFragmentSMEM*>(globalOutputs[globalWarpID*64])) PreAggregationHashtableFragmentSMEM(typeSize, scracthPad, scracthPadSize);
    }
    __syncwarp(); // Read allocated pointer after we are sure a thread has stored the allocation.
    PreAggregationHashtableFragmentSMEM* myFrag = reinterpret_cast<PreAggregationHashtableFragmentSMEM*>(globalOutputs[globalWarpID*64]);
    // if(!warpLane){
    //     printf("[TID=%d, WARPID=%d] myFrag=%p\n", globalTid, warpId, myFrag);
    // }

    // iterate over probe cols
    int probeColIdxStart = globalTid;
    int probeColIdxStep = numThreadsTotal;
    int roundedSize = ((probeColsLength + 31) / 32) * 32; 
    for(int probeColIdx = probeColIdxStart; probeColIdx < roundedSize; probeColIdx+=probeColIdxStep){
        bool remainInLoop{probeColIdx < probeColsLength};
        const int maskLoop = __ballot_sync(0xFFFFFFFF, remainInLoop);
        if(!remainInLoop){break;}
        ////// PROBE JOIN CONDITION //////
        const int key = probeCols[0][probeColIdx];
        const int val = probeCols[numProbeCols-1][probeColIdx];
        const uint32_t hash = hashInt32(key);
        const uint32_t pos = hash & view->htMask;
        HashIndexedViewEntry* current = reinterpret_cast<HashIndexedViewEntry*>(view->ht[pos]); // we have one view here (can have more in case of joins)
        bool foundMatch{false};
        while(current){ 
            foundMatch = (current->hashValue == hash && current->key == key);
            if (foundMatch) {break;}
            current = current->next;
        }
        ////// [END] PROBE JOIN CONDITION //////
        ////// INSERT/UPDATE PARTIAL AGGREGATE //////
        const int maskFoundMatch = __ballot_sync(maskLoop, foundMatch);
        if(foundMatch){
            const int groupVal = current->value;
            const uint32_t hashGroupCol = hashInt32(groupVal);
            const int scracthPadPos = (hashGroupCol >> htShift) & scracthPadMask;
            HashIndexedViewEntry* partialAggEntry = reinterpret_cast<HashIndexedViewEntry*>(scracthPad[scracthPadPos]);
            bool needInsert{true};
            if(!partialAggEntry){ 
                needInsert = true; // if no entry found (nullptr) at position
            } else {
                if(partialAggEntry->hashValue == hashGroupCol){ 
                    if(partialAggEntry->key == groupVal){
                        atomicAdd(&buildStats.hits, 1);
                        needInsert = false; // if found entry, hash and key match
                    } else { 
                        atomicAdd(&buildStats.keyMismatch, 1);
                        needInsert = true; // if key doesn't match, collision -> insert
                    }
                } else { 
                    atomicAdd(&buildStats.hashMismatch,1);
                    needInsert = true; // if hash doesn't match
                }
            }
            const int maskNeedInsert = __ballot_sync(maskFoundMatch, needInsert);
            if(needInsert){ // myFrag is warp-local, so multiple threads of the same warp can call insert()

                // printf("[%d]SETTING KEY %d of %p\n",threadIdx.x, groupVal,partialAggEntry);
                partialAggEntry = reinterpret_cast<HashIndexedViewEntry*>(myFrag->insert(hashGroupCol, maskNeedInsert));
                partialAggEntry->key = groupVal; // write key
                partialAggEntry->value = 0; // initialize value (aggregate)
                if(atomicExch((unsigned long long*)&myFrag->ht[scracthPadPos], (unsigned long long)partialAggEntry) == 0){atomicAdd(&buildStats.evicted, 1);} // only write ptr to an initialized entry 
            }
            atomicAdd(&partialAggEntry->value, val);
            // if(groupVal != outputEntry->value){
                // printf("[WarpId %d | TID %d] Key %d, Val %d\n", warpId, globalTid, groupVal, outputEntry->value);
            // }
        }
        ////// [END] INSERT/UPDATE PARTIAL AGGREGATE //////
    }
    __syncthreads();

    // if(!warpLane){ // DEBUG PRINT (for singlewarp/singlethreaded use)
    //     acquire_lock(&globalLock);
    //     myFrag->print(printEntry);
    //     release_lock(&globalLock);
    // }
    if(!warpLane){
        int su = 0;
        for(int i = 0; i < 64; i++){
            if(myFrag->outputs[i]){
                atomicAdd((unsigned long long*)&htSizes[i], (unsigned long long)myFrag->outputs[i]->getLen());
                su += myFrag->outputs[i]->getLen();
            }
        }
        memcpy(&globalOutputs[globalWarpID*64], myFrag->outputs, sizeof(FlexibleBuffer *) * 64);
        freePtr(myFrag);
    }
    ///////////////////////////////////////////
    // if(!warpLane){
    //     printf("[%d | WarpId %d] preAggrHTFrag length: %lu\n", blockIdx.x, warpId, myPreAggrHTFrag->len);
    // }
    // if(!threadIdx.x){ // check that preAggrHT is initialized
    //     buildStats.print();
    // }
}  



struct Content{
    bool flag;
    int32_t key;
    int32_t aggr;
};
__device__ bool eqInt(uint8_t* lhs, uint8_t* rhs){
    auto* lhsC = reinterpret_cast<Content*>(lhs);
    auto* rhsC = reinterpret_cast<Content*>(rhs);
    return lhsC->key == rhsC->key;
}
__device__ void combineInt(uint8_t* lhs, uint8_t* rhs){
    auto* lhsC = reinterpret_cast<Content*>(lhs);
    auto* rhsC = reinterpret_cast<Content*>(rhs);
    lhsC->aggr += rhsC->aggr;
}

__global__ void mergePreAggregationHashtableFragments(PreAggregationHashtable* preAggrHT, PreAggregationHashtable::PartitionHt* preAllocatedPartitions, FlexibleBuffer** globalOutputsVec, size_t numFrags, bool (*eq)(uint8_t*, uint8_t*), void (*combine)(uint8_t*, uint8_t*)) {
    const int warpCount = (blockDim.x + (warp_size-1)) / warp_size;
    const int warpId = threadIdx.x / warp_size;
    const int warpLane = threadIdx.x % warp_size;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    if(!globalTid){
        acquire_lock(&globalLock);
        new(preAggrHT) PreAggregationHashtable(preAllocatedPartitions);
        release_lock(&globalLock);

        // preAggrHT->print();
    } else if(!threadIdx.x){ // let each block wait until we touch the PreAggregationHashtable.
        acquire_lock(&globalLock);
        release_lock(&globalLock);
    }
    __syncthreads();
    int cntr{0};
    int numPartitions = 64;
    const int myPartitionId = blockIdx.x % numPartitions;
    const int blocksPerPartition = max(1,gridDim.x / numPartitions);
    const int blockIdxForPartition = blockIdx.x / numPartitions;


    const int fragmentStart = blockIdxForPartition;
    const int fragmentStep = blocksPerPartition;

    const int bufferElemStart = threadIdx.x;
    const int bufferElemStep = blockDim.x;
    // printf("{myPartitionId=%d, blocksPerPartition=%d, blockIdxForPartition=%d}\n", myPartitionId,blocksPerPartition,blockIdxForPartition);
    // printf("{fragmentStart=%d, fragmentStep=%d, bufferElemStart=%d, bufferElemStep=%d}\n", fragmentStart,fragmentStep,bufferElemStart,bufferElemStep);
    for(int i = 0; i < 64; i++){
    PreAggregationHashtable::Entry** myHt = preAggrHT->ht[i].ht;
    const size_t myHtMask = preAggrHT->ht[i].hashMask;

    for(int fragmentId = fragmentStart; fragmentId < numFrags; fragmentId += fragmentStep) {
        FlexibleBuffer* fragmentPartitionBuffer = globalOutputsVec[fragmentId * numPartitions + myPartitionId];
        if(!fragmentPartitionBuffer) { continue; }
        for(int bufferIdx = 0; bufferIdx < fragmentPartitionBuffer->buffers.count; bufferIdx++){
            Buffer* buf = &fragmentPartitionBuffer->buffers.payLoad[bufferIdx];
            for (int elementIdx = bufferElemStart; elementIdx < buf->numElements; elementIdx+=bufferElemStep) {
                PreAggregationHashtableFragment::Entry* curr = reinterpret_cast<PreAggregationHashtableFragment::Entry*>(&buf->ptr[elementIdx * typeSize]);
                const size_t pos = curr->hashValue >> PreAggregationHashtableFragment::htShift & myHtMask;
                PreAggregationHashtable::Entry* currCandidate = untag(myHt[pos]);
                bool merged = false;
                while (currCandidate) {
                    if (currCandidate->hashValue == curr->hashValue && eqInt(currCandidate->content, curr->content)) {
                        combineInt(currCandidate->content, curr->content);
                        merged = true;
                        break;
                    }
                    currCandidate = currCandidate->next;
                }
                if (!merged) {
                    // PreAggregationHashtable::Entry** loc = reinterpret_cast<PreAggregationHashtable::Entry**>(localBuffer.insert());
                    // *loc = curr;
                    PreAggregationHashtable::Entry* previousPtr = myHt[pos];
                    myHt[pos] = tag(curr, previousPtr, curr->hashValue);
                    myHt[pos] = curr;
                    curr->next = untag(previousPtr);
                } 
            }
        }
    }
    }
}

__global__ void INITPreAggregationHashtableFragmentsSingleThread(PreAggregationHashtable* preAggrHT, PreAggregationHashtable::PartitionHt* preAllocatedPartitions){
    if(blockDim.x * blockIdx.x + threadIdx.x == 0){
        new(preAggrHT) PreAggregationHashtable(preAllocatedPartitions);
    }
}
__global__ void mergePreAggregationHashtableFragmentsSingleThread(PreAggregationHashtable* preAggrHT, PreAggregationHashtable::PartitionHt* preAllocatedPartitions, FlexibleBuffer** globalOutputsVec, size_t numFrags) {
    const int warpCount = (blockDim.x + (warp_size-1)) / warp_size;
    const int warpId = threadIdx.x / warp_size;
    const int warpLane = threadIdx.x % warp_size;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;

    int cntr{0};
    /*
        - Partitions: have hts that are mutually exclusive in terms of sync -> partition-to-SM
        - Fragments: 
    */
    int partitionId = blockIdx.x % 64;
    int partitionWorkerId = blockIdx.x / 64;

    int blocks_per_partition = gridDim.x / 64;
    int extra_blocks = gridDim.x % 64;
    int stride = blocks_per_partition + (partitionId <= extra_blocks);

    PreAggregationHashtable::Entry** ht = preAggrHT->ht[partitionId].ht;
    const size_t htMask = preAggrHT->ht[partitionId].hashMask;
    __syncthreads();
    for(int fragmentId = 0; fragmentId < static_cast<int>(numFrags); fragmentId++){ 
        FlexibleBuffer* fragmentPartitionBuffer = globalOutputsVec[fragmentId * 64 + partitionId];
        if(!fragmentPartitionBuffer){continue;}
        for(int bufferIdx = partitionWorkerId; bufferIdx < fragmentPartitionBuffer->buffers.count; bufferIdx+=stride){
            Buffer* buf = &fragmentPartitionBuffer->buffers.payLoad[bufferIdx];
            for (int elementIdx = threadIdx.x; elementIdx < buf->numElements; elementIdx+=blockDim.x) {
                PreAggregationHashtableFragment::Entry* curr = reinterpret_cast<PreAggregationHashtableFragment::Entry*>(&buf->ptr[elementIdx * typeSize]);
                const size_t pos = curr->hashValue >> PreAggregationHashtableFragment::htShift & htMask;
                
                // auto* p = reinterpret_cast<HashIndexedViewEntry*>(curr);
                // printf("[Partition %d][POS %lu] MERGING hash=%llu, key=%d\n", partitionId, pos, p->hashValue, p->key);
                // PreAggregationHashtable::Entry* currCandidate = untag(ht[pos]);
                PreAggregationHashtable::Entry* currCandidate;
                do{
                    currCandidate = reinterpret_cast<PreAggregationHashtable::Entry*>(atomicExch((unsigned long long*)&ht[pos], 1ull));
                }
                while((unsigned long long)currCandidate == 1ull);

                bool merged = false;
                while (currCandidate) {
                    if (currCandidate->hashValue == curr->hashValue && eqInt(currCandidate->content, curr->content)) {
                        combineInt(currCandidate->content, curr->content);
                        merged = true;
                        break;
                    }
                    currCandidate = currCandidate->next;
                }
                if (!merged) {
                    PreAggregationHashtable::Entry* previousPtr = currCandidate;
                    currCandidate = tag(curr, previousPtr, curr->hashValue);
                    currCandidate = curr;
                    curr->next = untag(previousPtr);
                }
                atomicExch((unsigned long long*)&ht[pos], (unsigned long long)currCandidate);
                // if(atomicCAS((unsigned long long*)&ht[pos], 1ull, (unsigned long long)currCandidate) != 1ull){
                    // printf("Trouble\n");
                // }
            }
        }

    }
    // acquire_lock(&preAggrHT->mutex);
    // // Append buffers that back partition's pointers (no invalidation, because buffer itself is not reallocated)
    // preAggrHT->buffer.merge(localBuffer); 
    // release_lock(&preAggrHT->mutex);
}



__global__ void checkHashIndexSize(GrowingBuffer* buffer, HashIndexedView* view){
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    __shared__ int counter;
    if(threadIdx.x == 0){
        counter = 0;
    }
    __syncthreads();
    for (size_t i = globalTid; i < view->htMask+1; i+=numThreadsTotal) {
        HashIndexedView::Entry* current = view->ht[i];
        while (current != nullptr) {
            // printf("PTR %p, next %p\n", current, current->next);
            assert((unsigned long long)current != (unsigned long long)current->next);
            atomicAdd(&counter, 1);
            current = current->next;
        }
    }
    __syncthreads();
    if(threadIdx.x == 0){
        printf("Growing Buffer has %lu entries, HashIndexedView has %d entries\n", buffer->getLen(), counter);
        assert(buffer->getLen() == counter);
    }
}



// 8 mil: 10ms
// 1 mil: 1ms
int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " arraySizeElems numBlocks numThreadsInBlock numPredColumns printHeader(optional)\n";
        return 1;
    }

    int arraySizeElems = std::atoi(argv[1]);
    int numBlocks = std::atoi(argv[2]);
    int numThreadsInBlock = std::atoi(argv[3]);
    int numPredColumns = std::atoi(argv[4]);
    assert(numPredColumns && "Can't do 0 columns test");
    int printHeader = 1;
    if(argc == 6){
        printHeader = std::atoi(argv[5]);
    }

    #ifdef GALLATIN_ENABLED
    gallatin::allocators::init_global_allocator(heapSize, 10, false);
    #else
    hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize);
    #endif

    srand(10);
    const size_t allocSize = arraySizeElems * sizeof(int);
    int* hostCols[numPredColumns];
    int* devCols[numPredColumns];
    int** d_input_cols;
    CHECK_CUDA_ERROR(hipMalloc(&d_input_cols, sizeof(int*) * numPredColumns));
    CHECK_CUDA_ERROR(hipHostMalloc(&hostCols[0], allocSize));
    std::unordered_map<int, int> aggregated;
    for (int i = 0; i < arraySizeElems; ++i) {
        hostCols[0][i] = rand() % numbersThreshold;
        if(hostCols[0][i] < LTPredicate){
            aggregated[hostCols[0][i]] += hostCols[0][i];
        }
    }
    if(numbersThreshold==10){
        for(auto[key, value]: aggregated){
            std::cout << "{KEY: " << key << ", AGG: " << value << "}" << "\n";
        }
    }

    // If you use one PreAggregationHashtableFragment and one thread, it should return arraySizeElems - duplicateCount, keys with duplicates are aggregated.
    checkForDuplicates(hostCols[0], arraySizeElems, false); 
    for(int colidx = 0; colidx < numPredColumns; colidx++){
        CHECK_CUDA_ERROR(hipMalloc(&devCols[colidx], allocSize));
        CHECK_CUDA_ERROR(hipMemcpy(devCols[colidx], hostCols[0], allocSize, hipMemcpyHostToDevice));
    }
    CHECK_CUDA_ERROR(hipMemcpy(d_input_cols, devCols, sizeof(int*) * numPredColumns, hipMemcpyHostToDevice));


    int trueOutSize{0};
    for (int i = 0; i < arraySizeElems; ++i) {
        bool pred{true};
        if(pred){
            pred &= (hostCols[0][i] < LTPredicate);
        }
        trueOutSize += pred;
    } 
    // printf("TRUE OUT: %d\n", trueOutSize);

    GrowingBuffer* h_result;
    CHECK_CUDA_ERROR(hipHostMalloc(&h_result, sizeof(GrowingBuffer)));
    GrowingBuffer* result;
    CHECK_CUDA_ERROR(hipMalloc(&result, sizeof(GrowingBuffer)));

    HashIndexedView* h_result_view;
    CHECK_CUDA_ERROR(hipHostMalloc(&h_result_view, sizeof(HashIndexedView)));
    HashIndexedView* result_view;
    CHECK_CUDA_ERROR(hipMalloc(&result_view, sizeof(HashIndexedView)));

    PreAggregationHashtableFragment* h_result_preAggrHTFrag;
    CHECK_CUDA_ERROR(hipHostMalloc(&h_result_preAggrHTFrag, sizeof(PreAggregationHashtableFragment)));
    PreAggregationHashtableFragment* result_preAggrHTFrag;
    CHECK_CUDA_ERROR(hipMalloc(&result_preAggrHTFrag, sizeof(PreAggregationHashtableFragment)));


    int numWarps = getNumberOfElementsInSMEM(sizeof(PreAggregationHashtableFragment));
    size_t numThreadsInBlockPreAggr = 512; std::min(numThreadsInBlock, numWarps*32);
    std::cout << "[PreAggregationHashtableFragment] launch threads per block : " << numThreadsInBlockPreAggr 
        << ", SMEM can fit "<< numWarps*32 << " threads, SMEM size is " << getSharedMemorySize() << "B, PreAggregationHashtableFragment size is " << sizeof(PreAggregationHashtableFragment) << "B\n";
    PreAggregationHashtable* h_result_preAggrHT;
    CHECK_CUDA_ERROR(hipHostMalloc(&h_result_preAggrHT, sizeof(PreAggregationHashtable)));
    PreAggregationHashtable* result_preAggrHT;
    CHECK_CUDA_ERROR(hipMalloc(&result_preAggrHT, sizeof(PreAggregationHashtable)));

    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    const size_t sharedMemSizeGrowingBuf = sizeof(GrowingBuffer) * (32); // + sizeof(int) * 32 + sizeof(Buffer);
    const size_t sharedMemSizePreAggrHT = sizeof(PreAggregationHashtableFragment) * (2); // + sizeof(int) * 32 + sizeof(Buffer);

    if(printHeader){
        printf("Kernel type,Num cols,Init buffer size,Num bytes,Num Blocks,Num threads,GrowingBuffer Time,HashIndexedView Time,Malloc Count,Kernel malloc,Vec malloc,Next buf malloc,Free,Result total len\n");
    }
    using KernelFuncPtr = void (*)(int**, int, int, GrowingBuffer*);
    const size_t numRuns{1};
    float timeMs_GrowingBuffer = 0.0f;
    float timeMs_HashIndexedView = 0.0f;

    std::cout << "Launch config: <<<" <<numBlocks << ","<<numThreadsInBlock <<  ">>>\n";
    auto runMallocBench = [&](KernelFuncPtr funcPtr, const std::string& name){
        timeMs_HashIndexedView = 0.0;
        timeMs_GrowingBuffer=0.0;
        for(int i = 0; i < numRuns+1; i++){
            memset(counters, 0, 4*sizeof(int));
            CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(deviceCounters), counters, 4 * sizeof(int), 0, hipMemcpyHostToDevice));
            hipEventRecord(start, 0);
            //////////////// Build GrowingBuffer ////////////////
            growingBufferInit<<<1,1>>>(result);
            hipDeviceSynchronize();

            growingBufferFillTB<<<numBlocks, numThreadsInBlock, sharedMemSizeGrowingBuf>>>(d_input_cols, numPredColumns, arraySizeElems, result);
            hipDeviceSynchronize();
            //////////////// Build GrowingBuffer ////////////////
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float milliseconds = 0.0;
            hipEventElapsedTime(&milliseconds, start, stop);
            timeMs_GrowingBuffer += (i > 0) ? milliseconds : 0.0;

            auto t = hipGetLastError();
            CHECK_CUDA_ERROR(t);

            hipEventRecord(start, 0);
            //////////////// Build HashIndexedView ////////////////
            CHECK_CUDA_ERROR(hipMalloc(&result_view, sizeof(HashIndexedView))); // device-hosted HashIndexedView
            CHECK_CUDA_ERROR(hipMemcpy(h_result, result, sizeof(GrowingBuffer), hipMemcpyDeviceToHost)); // need to calculate ht size

            auto [htAllocSize, htMask] = getHtSizeMask(h_result->getValues().getLen(), sizeof(HashIndexedView::Entry*));
            h_result_view->htMask = htMask;
            CHECK_CUDA_ERROR(hipMalloc(&h_result_view->ht, htAllocSize)); // set ht pointer
            CHECK_CUDA_ERROR(hipMemset(h_result_view->ht, 0, htAllocSize)); // zero ht buffer

            CHECK_CUDA_ERROR(hipMemcpy(result_view, h_result_view, sizeof(HashIndexedView), hipMemcpyHostToDevice)); // copy ht pointer and ht mask

            int hashBuilderNumBlocks=h_result->getValues().buffers.count;
            std::cout << "[HashIndexedView] htSize="<< htAllocSize << "B, h_result->getValues().getLen()= " << h_result->getValues().getLen() << ", h_result->getValues().buffers.count=" << h_result->getValues().buffers.count << "\n";
            int hashBuilderNumThreadsPerTB=((std::min(h_result->getValues().getLen()/hashBuilderNumBlocks, 1024) +31)/32) *32;
            std::cout << "[buildHashIndexedView] Launch config: <<<" <<hashBuilderNumBlocks << ","<<hashBuilderNumThreadsPerTB <<  ">>>\n";
            // buildHashIndexedViewAdvancedSMEMWarpLevel<HashIndexedViewBuilderType::BufferToSM><<<hashBuilderNumBlocks,256>>>(result, result_view);
            // CHECK_CUDA_ERROR(hipMemset(h_result_view->ht, 0, htAllocSize)); // zero ht buffer
            buildHashIndexedViewAdvancedSMEM<HashIndexedViewBuilderType::BufferToSM><<<hashBuilderNumBlocks,256>>>(result, result_view);
            CHECK_CUDA_ERROR(hipMemset(h_result_view->ht, 0, htAllocSize)); // zero ht buffer
            buildHashIndexedView<HashIndexedViewBuilderType::BufferToSM><<<numBlocks,numThreadsInBlock>>>(result, result_view);
            // printHashIndexedView<<<1,1>>>(result_view);
            hipDeviceSynchronize();
            t = hipGetLastError();
            CHECK_CUDA_ERROR(t);
            //////////////// Build HT FRAGMENTS ////////////////
            FlexibleBuffer** allOutputs_d; // store warp-level fragment's partition's FlexibleBuffers (X warps * 64 partitions)
            size_t* outputsSizes_d; // store partition sizes
            size_t* outputsSizes_h;
            CHECK_CUDA_ERROR(hipHostMalloc(&outputsSizes_h, sizeof(size_t) * 64));
            CHECK_CUDA_ERROR(hipMalloc(&outputsSizes_d, sizeof(size_t) * 64));
            hipMemset(outputsSizes_d, 0, sizeof(size_t) * 64); // sizes are accumulated, so first init to 0 

            const size_t numFragments = max(1ul,(numBlocks*numThreadsInBlockPreAggr)/32); // each warp has 1 fragment

            const size_t outputsPointersArraySize = sizeof(FlexibleBuffer*) * (64 * numFragments); // each fragment has 64 partitions

            CHECK_CUDA_ERROR(hipMalloc(&allOutputs_d, outputsPointersArraySize)); 
            std::cout << "[buildPreAggregationHashtableFragments] Launch config: <<<" <<numBlocks << ","<<numThreadsInBlockPreAggr <<  ">>>\n";
            // buildPreAggregationHashtableFragments<<<numBlocks,numThreadsInBlockPreAggr,getSharedMemorySize()>>>(d_input_cols, numPredColumns, arraySizeElems, result_view, allOutputs_d, outputsSizes_d);
            //
            // 1024-128 due to high register usage
            buildPreAggregationHashtableFragmentsAdvanced<<<numBlocks,256>>>(d_input_cols, numPredColumns, arraySizeElems, result_view, allOutputs_d, outputsSizes_d);
            hipDeviceSynchronize();
            t = hipGetLastError();
            CHECK_CUDA_ERROR(t);
            //////////////// Merge HT FRAGMENTS ////////////////
            CHECK_CUDA_ERROR(hipMemcpy(outputsSizes_h, outputsSizes_d, sizeof(size_t) * 64, hipMemcpyDeviceToHost)); // get sizes back
            size_t totalSum{0}; // debug
            PreAggregationHashtable::PartitionHt* preAllocatedPartitions_d;
            CHECK_CUDA_ERROR(hipMalloc(&preAllocatedPartitions_d, sizeof(PreAggregationHashtable::PartitionHt) * 64));
            PreAggregationHashtable::PartitionHt* preAllocatedPartitions_h;
            CHECK_CUDA_ERROR(hipHostMalloc(&preAllocatedPartitions_h, sizeof(PreAggregationHashtable::PartitionHt) * 64));

            for(int outputId = 0; outputId < PreAggregationHashtableFragment::numOutputs; outputId++){ // allocate ht buffer for each final partition
                auto [htAllocSize, htMask] = getHtSizeMask(outputsSizes_h[outputId], sizeof(PreAggregationHashtableFragment::Entry*));
                preAllocatedPartitions_h[outputId].hashMask = htMask;
                CHECK_CUDA_ERROR(hipMalloc(&preAllocatedPartitions_h[outputId].ht, htAllocSize));
                CHECK_CUDA_ERROR(hipMemset(preAllocatedPartitions_h[outputId].ht, 0, htAllocSize));
                // totalSum += outputsSizes_h[outputId];
                // std::cout << "[HOST][ALLOCATE FINAL HT PARTITION "<< outputId << "], given size " << outputsSizes_h[outputId] << ": allocSize=" << htAllocSize << " B, at " << preAllocatedPartitions_h[outputId].ht << ", ht mask is " << htMask << "\n";
            }
            // std::cout << "[Merge HT FRAGMENTS] Total size = " << totalSum << ", num fragments " << numFragments << "\n";
            CHECK_CUDA_ERROR(hipMemcpy(preAllocatedPartitions_d, preAllocatedPartitions_h, sizeof(PreAggregationHashtable::PartitionHt) * 64, hipMemcpyHostToDevice));
            // dim3 dimBlock(64);  // Cooperative launch, use (1, 1, 1) for block dimensions
            // dim3 dimGrid(64);   // Set grid dimensions according to your problem size

            // // Prepare kernel arguments
            // void* kernelArgs[] = {
            //     &result_preAggrHT,
            //     &preAllocatedPartitions_d,
            //     &allOutputs_d,
            //     (void*)&numFragments
            // };

            // // Launch the kernel using hipLaunchCooperativeKernel
            // hipError_t err = hipLaunchCooperativeKernel(
            //     (void*)mergePreAggregationHashtableFragmentsSingleThread,  // Kernel function pointer
            //     dimGrid,    // Grid dimensions
            //     dimBlock,   // Block dimensions
            //     kernelArgs  // Array of kernel arguments
            // );
            // hipDeviceSynchronize();
            INITPreAggregationHashtableFragmentsSingleThread<<<1,1>>>(result_preAggrHT, preAllocatedPartitions_d);
            hipDeviceSynchronize();

            mergePreAggregationHashtableFragmentsSingleThread<<<256,512>>>(result_preAggrHT, preAllocatedPartitions_d, allOutputs_d, numFragments);
            hipDeviceSynchronize();
            if(numbersThreshold==10){
                printPreAggregationHashtable<<<1,1>>>(result_preAggrHT, false);
            }
            hipDeviceSynchronize();
            t = hipGetLastError();
            CHECK_CUDA_ERROR(t);
            //////////////////////////////////////////////////////
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            milliseconds=0.0;
            hipEventElapsedTime(&milliseconds, start, stop);
            timeMs_HashIndexedView += (i > 0) ? milliseconds : 0.0;

            t = hipGetLastError();
            CHECK_CUDA_ERROR(t);

            // timeMs += milliseconds;
            checkHashIndexSize<<<1, 1024>>>(result, result_view);
            freeKernel<<<1, 1>>>(result, result_view);
            CHECK_CUDA_ERROR(hipFree(h_result_view->ht));

            hipDeviceSynchronize();
            t = hipGetLastError();
            CHECK_CUDA_ERROR(t);


            for(int outputId = 0; outputId < PreAggregationHashtableFragment::numOutputs; outputId++){
                void* htOutput = h_result_preAggrHT->ht[outputId].ht;
                CHECK_CUDA_ERROR(hipFree(htOutput));
            }
            hipFree(allOutputs_d);
            hipFree(outputsSizes_d);

            hipHostFree(outputsSizes_h);
        }
        CHECK_CUDA_ERROR(hipMemcpyFromSymbol(counters, HIP_SYMBOL(deviceCounters), 4 * sizeof(int), 0, hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipMemcpy(h_result, result, sizeof(GrowingBuffer), hipMemcpyDeviceToHost));
        printf("%s,%d,%d,%lu,%d,%d,%.3f,%.3f,%d,%d,%d,%d,%d,%d\n", 
            name.c_str(), numPredColumns, initialCapacity, allocSize, numBlocks, numThreadsInBlock, timeMs_GrowingBuffer/numRuns,timeMs_HashIndexedView/numRuns,  
            counters[(int)Counter::InitBufferMalloc]+counters[static_cast<int>(Counter::NextBufferMalloc)]+counters[static_cast<int>(Counter::VectorExpansionMalloc)],
            counters[static_cast<int>(Counter::InitBufferMalloc)], counters[static_cast<int>(Counter::VectorExpansionMalloc)], 
            counters[static_cast<int>(Counter::NextBufferMalloc)], counters[static_cast<int>(Counter::Free)], h_result->getValues().getLen());
        // free(mallodb);  
    };

    #ifdef GALLATIN_ENABLED
    runMallocBench(growingBufferFill, "WarpLevel (Gallatin)");
    #else
    runMallocBench(growingBufferFill, "WarpLevel");
    #endif

    // Free memory
    for (int colidx = 0; colidx < numPredColumns; ++colidx) {
        CHECK_CUDA_ERROR(hipFree(devCols[colidx]));
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipDeviceReset();
    return 0;
}

