#include "hip/hip_runtime.h"
#include "gpu_utils.h"
#include "ssb_utils.h"

#include "GrowingBuffer.cuh"
#include "LazyJoinHashtable.cuh"
#include "PreAggregationHashtable.cuh"
#include "PrefixSum.cuh"
#include "helper.cuh"
#include <hip/hip_runtime.h>

constexpr uint64_t KiB = 1024;
constexpr uint64_t MiB = 1024 * KiB;
constexpr uint64_t GiB = 1024 * MiB;
constexpr uint64_t HEAP_SIZE{3*GiB};
constexpr uint64_t NUM_RUNS = 2;

int sf=1;

// ./Bench /somepath/crystal/test/ssb/data/ 1 1

constexpr int INIT_CAPACITY = INITIAL_CAPACITY;
constexpr int WARP_SIZE = 32;

std::string h_DATA_DIR;
int h_LO_LEN;
int h_P_LEN;
int h_S_LEN;
int h_C_LEN;
int h_D_LEN;
__constant__ int d_LO_LEN;
__constant__ int d_P_LEN;
__constant__ int d_S_LEN;
__constant__ int d_C_LEN;
__constant__ int d_D_LEN;

void initialize(int sf) {
    switch (sf) {
        case 1:
            h_DATA_DIR = "s1_columnar/";
            h_LO_LEN = 6001171;
            h_P_LEN = 200000;
            h_S_LEN = 2000;
            h_C_LEN = 30000;
            h_D_LEN = 2556;
            break;
        case 10:
            h_DATA_DIR = "s10_columnar/";
            h_LO_LEN = 59986214;
            h_P_LEN = 800000;
            h_S_LEN = 20000;
            h_C_LEN = 300000;
            h_D_LEN = 2556;
            break;
        // Add more cases if needed
        default:
            std::cerr << "Unsupported SF value: " << SF << std::endl;
            exit(1);
    }
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_LO_LEN), &h_LO_LEN, sizeof(int), 0, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_P_LEN), &h_P_LEN, sizeof(int), 0, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_S_LEN), &h_S_LEN, sizeof(int), 0, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_C_LEN), &h_C_LEN, sizeof(int), 0, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_D_LEN), &h_D_LEN, sizeof(int), 0, hipMemcpyHostToDevice));
}

//////////////////////////////////////////////// QUERY 4.1 ////////////////////////////////////////////////

enum class TABLE{S = 0, C = 1, P = 2, D = 3};

// all columns scan one key and one val. If needed, specialize to GrowingBufEntryScan*COLNAME* 
struct GrowingBufEntryScan { 
    GrowingBufEntryScan* next;
    uint64_t hashValue;
    bool nullFlag;
    int32_t key; // e.g., lo_orderdate or lo_partkey
    int32_t value; // e.g., d_year or c_nation
};

__device__ void printEntryScan(uint8_t* entryPtr){
    GrowingBufEntryScan* structPtr = reinterpret_cast<GrowingBufEntryScan*>(entryPtr);
    printf("{key=%d,val=%d,hash=%llu,next=%p},", structPtr->key, structPtr->value, structPtr->hashValue, structPtr->next);
}

struct GrowingBufEntryResHT {
    GrowingBufEntryResHT* next;
    uint64_t hashValue;
    bool nullFlag;
    int32_t key[2]; // d_year,c_nation ... group by d_year,c_nation
    int64_t value; // (Q4.1 sum(lo_revenue-lo_supplycost) as profit)
};

__device__ void printEntryResHT(uint8_t* entryPtr){
    GrowingBufEntryResHT* structPtr = reinterpret_cast<GrowingBufEntryResHT*>(entryPtr);
    printf("{key_1=%d,key_2=%d,val=%lld,hash=%llu,next=%p},", structPtr->key[0], structPtr->key[1], structPtr->value, structPtr->hashValue, structPtr->next);
}

constexpr uint64_t TYPE_SIZE_SCAN{sizeof(GrowingBufEntryScan)}; // all columns scan one key and one val. If needed, specialize to TYPE_SIZE_SCAN_*COLNAME* 
constexpr uint64_t TYPE_SIZE_RES_HT{sizeof(GrowingBufEntryResHT)};

std::pair<size_t, size_t> getHtSizeMask(size_t numElements, size_t elementSize){
    size_t size = max(PreAggregationHashtable::nextPow2(numElements * 1.25), 1ull);
    return {size*elementSize, size-1};
}

__device__ __forceinline__ uint64_t combineHashes(uint64_t hash1, uint64_t hash2) {
    return hash1 ^ (hash2 + 0x9e3779b9 + (hash1 << 6) + (hash1 >> 2));
}

__device__ __forceinline__ int64_t hashInt32ToInt64(int32_t x) {
    uint64_t prime = 0x9e3779b97f4a7c15;
    uint32_t ux = static_cast<uint32_t>(x);
    ux ^= (ux >> 30);
    uint64_t result = ux * prime;
    result ^= (result >> 27);
    result *= prime;
    result ^= (result >> 31);
    result = result & 0xFFFFFFFFFFFFFFFF;
    return static_cast<int64_t>(result);
}

__device__ __forceinline__ uint32_t hashInt32(int32_t key) {
    key ^= key >> 16;
    key *= 0x85ebca6b;
    key ^= key >> 13;
    key *= 0xc2b2ae35;
    key ^= key >> 16;
    return key;
}

__global__ void growingBufferInit(GrowingBuffer* finalBuffer) {
    if(blockDim.x * blockIdx.x + threadIdx.x == 0){
        new(finalBuffer) GrowingBuffer(INIT_CAPACITY, TYPE_SIZE_SCAN, false);
    }
}

__device__ volatile int GLOBAL_LOCK{0};

enum class FillVariant{
    ThreadBlockLockStep = 1,
    Opportunistic = 2
};

template<TABLE Table, FillVariant Impl = FillVariant::ThreadBlockLockStep>
__global__ void growingBufferFillTB(int* filterCol, int* keyCol, int* valueCol, int numTuples, GrowingBuffer* finalBuffer) {
    const int warp_count = (blockDim.x + (WARP_SIZE-1)) / WARP_SIZE;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    const int lane = threadIdx.x % WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;

    __shared__ char sharedMem[2048];
    LeafFlexibleBuffer* myBuf = reinterpret_cast<LeafFlexibleBuffer*>(sharedMem);
    GrowingBufEntryScan** writeCursor = reinterpret_cast<GrowingBufEntryScan**>(sharedMem + sizeof(LeafFlexibleBuffer));
    uint32_t* counter = reinterpret_cast<uint32_t*>(sharedMem + sizeof(LeafFlexibleBuffer) + sizeof(GrowingBufEntryScan**));

    if (threadIdx.x == 0) {
        new (myBuf) LeafFlexibleBuffer(INIT_CAPACITY, TYPE_SIZE_SCAN, false);
        *counter = 0;
    }
    __syncthreads();
    uint32_t myIdx = 0;

    int roundedSize = ((numTuples + (WARP_SIZE-1)) / WARP_SIZE) * WARP_SIZE;
    for (int i = globalTid; i < roundedSize; i += numThreadsTotal) {
        bool pred = (i < numTuples);
        if(pred){
            if constexpr(Table == TABLE::S){
                pred &= (filterCol[i] == 1);
            } else if constexpr (Table == TABLE::P){
                pred &= (filterCol[i] == 1 || filterCol[i] == 0);
            } else if constexpr (Table == TABLE::C){
                pred &= (filterCol[i] == 1);
            } else if constexpr (Table == TABLE::D){
                // No filter
            }
        }
        if constexpr(Impl == FillVariant::ThreadBlockLockStep){
            const int maskWriters = __ballot_sync(__activemask(), pred);
            const int leader = __ffs(maskWriters)-1;
            if(lane == leader){
                // _block ensures memory ordering only for this thread block (a more relaxed atomic)
                myIdx = atomicAdd_block(counter, __popc(maskWriters)); // Shared load, heavy inst
            }
            myIdx = __shfl_sync(maskWriters, myIdx, leader) + __popc(maskWriters & ((1U << lane) - 1)); // barrier stalls
            __syncthreads();
            if (threadIdx.x == 0) { // Critical section, try to reduce time (e.g., put myBuf in SMEM).
                *writeCursor = (GrowingBufEntryScan*)myBuf->insert(*counter);
                *counter = 0;
            }
            __syncthreads();
        }
        GrowingBufEntryScan* writeTo;
        if (pred) { // Uncoalesced stores
            if constexpr(Impl == FillVariant::ThreadBlockLockStep){
                writeTo = *writeCursor; // Shared load
            } else {
                writeTo = (GrowingBufEntryScan*)myBuf->insertWarpLevelOpportunistic(); // Shared load
            }
            writeTo[myIdx].key /*[0], [1] for many keys*/ = keyCol[i]; // Global 2 loads, 1 store (LG throttle: L2 can't keep up)
            writeTo[myIdx].hashValue = hashInt32ToInt64(keyCol[i]); // Global store
            if constexpr (Table == TABLE::D || Table == TABLE::C){
                writeTo[myIdx].value /*[0], [1] for many vals*/ = valueCol[i]; // Global 2 loads, 1 store (many stalls)
            }
        } 

    }
    __syncthreads();

    if (threadIdx.x == 0) {
        while (atomicCAS(&finalBuffer->getValuesPtr()->lock, 0, 1) != 0);
        finalBuffer->getValuesPtr()->merge(myBuf);
        atomicExch(&finalBuffer->getValuesPtr()->lock, 0);
    }
    // if(!threadIdx.x){finalBuffer->getValues().print(printEntryScan);}  // only for <<<1,X>>> debug
}

enum class HashIndexedViewBuilderType{
    BufferToSM = 1,
    BufferToGPU = 2
};

struct ViewCachedSubchain{
    HashIndexedView::Entry* head{nullptr};
    HashIndexedView::Entry* tail{nullptr};
    int64_t writeOutPos{-1}; 
};
__device__ void atomicAppendSubList(HashIndexedView::Entry** globalHt, const size_t pos, HashIndexedView::Entry* subListHead, HashIndexedView::Entry* subListTail) {
    HashIndexedView::Entry* currentHead = globalHt[pos];
    HashIndexedView::Entry* old = currentHead;
    const uint64_t hash = subListHead->hashValue; // global load
    do {
        currentHead = old;
        if(subListHead != subListTail){ // We set tail to head on the first write to head, if they do not match -> sub-list length > 1, need to adjust tail
            subListTail->next = currentHead; 
        } else // if subListHead is not different from tail -> sub-list length 1, only adjust head
            subListHead->next = currentHead; 
        subListHead = tag(subListHead, currentHead, hash);
        old = (HashIndexedView::Entry*) atomicCAS((unsigned long long*)&globalHt[pos], (unsigned long long)currentHead, (unsigned long long)subListHead);
    } while (old != currentHead);
}

__device__ __forceinline__ void atomicAppendSMEM(ViewCachedSubchain* ht, const size_t pos, HashIndexedView::Entry* newNode) {
    HashIndexedView::Entry* currentHead = ht[pos].head; // shared load
    HashIndexedView::Entry* old = currentHead;
    const uint64_t hash = newNode->hashValue; // global load (scoreboard stalls)
    do {
        currentHead = old;
        newNode->next = currentHead;  // global store
        newNode = tag(newNode, currentHead, hash);
        old = (HashIndexedView::Entry*) atomicCAS((unsigned long long*)&ht[pos].head, (unsigned long long)currentHead, (unsigned long long)newNode);
    } while (old != currentHead);
    if(!old) { // We executed the first write (meaning head was nullptr before): set tail to head. No other write to this head would update it anymore -> thread safe.
        ht[pos].tail = newNode;
    }
}

__device__ __forceinline__ void atomicAppend(HashIndexedView::Entry** globalHt, const size_t pos, HashIndexedView::Entry* newNode) {
    HashIndexedView::Entry* currentHead = globalHt[pos]; // global load (scoreboard stalls) inefficient memory access patterns 
    HashIndexedView::Entry* old = currentHead; 
    const uint64_t hash = newNode->hashValue; // global load (scoreboard stalls) inefficient memory access patterns 
    do {
        currentHead = old;
        newNode->next = currentHead; // global store (L2 throttle) inefficient memory access patterns 
        newNode = tag(newNode, currentHead, hash);
        old = (HashIndexedView::Entry*) atomicCAS((unsigned long long*)&globalHt[pos], (unsigned long long)currentHead, (unsigned long long)newNode);
    } while (old != currentHead); // takes many instructions (but few stalls)
}

template<HashIndexedViewBuilderType Qimpl = HashIndexedViewBuilderType::BufferToSM>
__global__ void buildHashIndexedViewAdvancedSMEM(GrowingBuffer* buffer, HashIndexedView* view) {
    const int warpCount = (blockDim.x + (WARP_SIZE-1)) / WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;
    const int warpLane = threadIdx.x % WARP_SIZE;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;

    const int powerTwoTemp{10};
    const size_t scracthPadMask{(1 << powerTwoTemp) - 1};
    __shared__ ViewCachedSubchain scracthPad[1 << powerTwoTemp];

    for(int i = threadIdx.x; i < scracthPadMask+1; i+=blockDim.x){
        scracthPad[i].writeOutPos = -1ll;
        scracthPad[i].head = nullptr;
        scracthPad[i].tail = nullptr;
    }
    __syncthreads();

    auto* values = buffer->getValuesPtr();
    int bufferIdxStart{0};
    int bufferIdxStep{0};
    int bufferEntryIdxStart{0};
    int bufferEntryIdxStep{0};
    if constexpr(Qimpl == HashIndexedViewBuilderType::BufferToSM){
        bufferIdxStart = blockIdx.x;
        bufferIdxStep = gridDim.x;
        bufferEntryIdxStart = threadIdx.x;
        bufferEntryIdxStep = blockDim.x;
    }
    else{
        bufferIdxStart = 0;
        bufferIdxStep = 1;
        bufferEntryIdxStart = globalTid;
        bufferEntryIdxStep = numThreadsTotal;
    }
    // int conflictCnt{0};
    const int buffersCnt{values->getBuffers().size()}; // Global load 
    const size_t globalMask{view->htMask}; // Global load 
    HashIndexedView::Entry** globalHt{view->ht}; // Global load 

    for(int bufIdx=bufferIdxStart; bufIdx<buffersCnt; bufIdx+=bufferIdxStep){  
        auto* buffer = &values->getBuffers()[bufIdx];
        const int entryCnt{buffer->numElements}; // Global load 
        for (int bufEntryIdx = bufferEntryIdxStart; bufEntryIdx < entryCnt; bufEntryIdx+=bufferEntryIdxStep) { 
            HashIndexedView::Entry* entry = (HashIndexedView::Entry*) &buffer->ptr[bufEntryIdx * TYPE_SIZE_SCAN]; // if needed, specialize TYPE_SIZE_SCAN
            size_t hash = (size_t) entry->hashValue; // Global load (heavy)
            const size_t posGlobal = hash & globalMask;
            HashIndexedView::Entry* newEntry;
            HashIndexedView::Entry* current;
            HashIndexedView::Entry* exchanged;
            do {
                current=globalHt[posGlobal];
                entry->next=current;
                newEntry = tag(entry, current, hash);
                exchanged = (HashIndexedView::Entry*) atomicCAS((unsigned long long*)&globalHt[posGlobal], (unsigned long long)current, (unsigned long long)newEntry);
            } while (exchanged!=current);
            /*
            const size_t posLocal = hash & scracthPadMask;
            const int64_t writeOutPos = atomicCAS((unsigned long long*)&scracthPad[posLocal].writeOutPos, (unsigned long long)-1, (unsigned long long)posGlobal);
            if(writeOutPos == -1 || writeOutPos == posGlobal){ // Was an empty SMEM slot, we just occupied it with posGlobal. OR matched the writeout position.
                atomicAppendSMEM(scracthPad, posLocal, entry);
            } else { // If we have a collision (scratch pad's entry writeout position != entry's writeOut pos) -> write directly to global.
                atomicAppend(globalHt, posGlobal, entry);
            }
            */
        }
    }
    __syncthreads();
    // for(int i = threadIdx.x; i < scracthPadMask+1; i+=blockDim.x){
    //     if(scracthPad[i].head){
    //         atomicAppendSubList(view->ht, scracthPad[i].writeOutPos, scracthPad[i].head, scracthPad[i].tail);
    //     }
    // }
    // if(!threadIdx.x){view->print();}  // only for <<<1,X>>> debug
}

struct ViewResult{
    GrowingBuffer* h_filter_scan{nullptr};
    GrowingBuffer* d_filter_scan{nullptr}; 
    HashIndexedView* h_hash_view{nullptr}; 
    HashIndexedView* d_hash_view{nullptr};
};

template<TABLE Table>
ViewResult buildView(int* filterCol, int* keyCol, int* valCol, int numTuples){
    ViewResult res;
    CHECK_CUDA_ERROR(hipHostMalloc(&res.h_filter_scan, sizeof(GrowingBuffer)));
    CHECK_CUDA_ERROR(hipMalloc(&res.d_filter_scan, sizeof(GrowingBuffer)));
    CHECK_CUDA_ERROR(hipHostMalloc(&res.h_hash_view, sizeof(HashIndexedView)));
    CHECK_CUDA_ERROR(hipMalloc(&res.d_hash_view, sizeof(HashIndexedView)));
    
    growingBufferInit<<<1,1>>>(res.d_filter_scan);
    // If you execute __syncthreads() to synchronize the threads of a block, it is recommended to have more than the achieved 1 blocks per multiprocessor. 
    // This way, blocks that aren't waiting for __syncthreads() can keep the hardware busy
    growingBufferFillTB<Table><<<30,1024>>>(filterCol, keyCol, valCol, numTuples, res.d_filter_scan); 
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR(hipGetLastError());


    CHECK_CUDA_ERROR(hipMemcpy(res.h_filter_scan, res.d_filter_scan, sizeof(GrowingBuffer), hipMemcpyDeviceToHost));
    auto [htAllocSize, htMask] = getHtSizeMask(res.h_filter_scan->getValuesPtr()->getLen(), sizeof(GrowingBufEntryScan*)); // if needed, specialize GrowingBufEntryScan*
    std::cout << "Filter in: " << numTuples << ", filter out: " <<  res.h_filter_scan->getValuesPtr()->getLen() << "\n";
    res.h_hash_view->htMask = htMask;
    CHECK_CUDA_ERROR(hipMalloc(&res.h_hash_view->ht, htAllocSize));
    CHECK_CUDA_ERROR(hipMemset(res.h_hash_view->ht, 0, htAllocSize));
    CHECK_CUDA_ERROR(hipMemcpy(res.d_hash_view, res.h_hash_view, sizeof(HashIndexedView), hipMemcpyHostToDevice));
    
    buildHashIndexedViewAdvancedSMEM<HashIndexedViewBuilderType::BufferToGPU><<<30,256>>>(res.d_filter_scan, res.d_hash_view);
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR(hipGetLastError());

    return res;
}

/*
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
__global__ void buildPreAggregationHashtableFragmentsAdvancedCG(
        int* lo_orderdate, int* lo_partkey, int* lo_custkey, int* lo_suppkey, int* lo_revenue, int* lo_supplycost, int lo_len,
        HashIndexedView* sView, HashIndexedView* cView, HashIndexedView* pView, HashIndexedView* dView, 
        FlexibleBuffer** globalOutputs, size_t* htSizes) 
    {
    //  1024 threads per block, the maximum registers per thread is 64
    const int warpCount = (blockDim.x + (WARP_SIZE-1)) / WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;
    const int warpLane = threadIdx.x % WARP_SIZE;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    const int globalWarpID = globalTid / WARP_SIZE;

    constexpr size_t outputMask = PreAggregationHashtableFragment::numOutputs - 1;
    constexpr size_t htShift = 6; 


    // SMEM size is very important for reducing work in the merge phase. 
    // Example RTX 2060, Q4.1. SF10: 2^10 leads to 10ms in merge, 2^12 leads to 170us-1ms(!) in merge.
    // However, the fragment building phase remains bottlenecked by the read latency (scan of a large relation with probes).
    const int powerTwoTemp{12}; 
    const int scracthPadSize{1 << powerTwoTemp};
    const size_t scracthPadMask{scracthPadSize - 1};

    // __shared__ PreAggregationHashtableFragmentSMEM::Entry* scracthPad[scracthPadSize];
    __shared__ char smem[
        scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*) 
        + 4*sizeof(HashIndexedView) 
        + sizeof(PreAggregationHashtableFragmentSMEM)];
    PreAggregationHashtableFragmentSMEM::Entry** scracthPad = reinterpret_cast<PreAggregationHashtableFragmentSMEM::Entry**>(smem);
    HashIndexedView* cachedView_S = reinterpret_cast<HashIndexedView*>(smem + scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*));
    HashIndexedView* cachedView_P = reinterpret_cast<HashIndexedView*>(smem + scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*) + sizeof(HashIndexedView));
    HashIndexedView* cachedView_D = reinterpret_cast<HashIndexedView*>(smem + scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*) + 2*sizeof(HashIndexedView));
    HashIndexedView* cachedView_C = reinterpret_cast<HashIndexedView*>(smem + scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*) + 3*sizeof(HashIndexedView));
    PreAggregationHashtableFragmentSMEM* myFrag = reinterpret_cast<PreAggregationHashtableFragmentSMEM*>(smem + scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*) + 4*sizeof(HashIndexedView));
    
    cg::thread_block block = cg::this_thread_block();
    for(int i = threadIdx.x; i < scracthPadSize; i+=blockDim.x){
        scracthPad[i] = nullptr;
    }
    if(threadIdx.x == 0){
        *cachedView_S = *sView;
        *cachedView_P = *pView;
        *cachedView_D = *dView;
        *cachedView_C = *cView;
        new(myFrag) PreAggregationHashtableFragmentSMEM(TYPE_SIZE_RES_HT, scracthPad, scracthPadSize);
    }
    block.sync();

    // iterate over probe cols
    int probeColIdxStart = globalTid;
    int probeColIdxStep = numThreadsTotal;
    int roundedSize = ((lo_len + 31) / 32) * 32; 
    for(int probeColTupleIdx = probeColIdxStart; probeColTupleIdx < roundedSize; probeColTupleIdx+=probeColIdxStep){
        cg::coalesced_group active = cg::coalesced_threads();
        const bool remainInLoop{probeColTupleIdx < lo_len};
        bool foundMatch{false}; // PROBING
        GrowingBufEntryScan* current_C{nullptr}; // value cols
        GrowingBufEntryScan* current_D{nullptr}; 
        if(remainInLoop){
        ////// PROBE S JOIN CONDITION //////
        const int lo_key_S = lo_suppkey[probeColTupleIdx];
        const uint32_t hash_S = hashInt32(lo_key_S);
        const size_t pos_S = hash_S & cachedView_S->htMask;
        GrowingBufEntryScan* current_S = reinterpret_cast<GrowingBufEntryScan*>(filterTagged(cachedView_S->ht[pos_S], hash_S)); // we have one view here (can have more in case of joins) // Global load (uncoalesced)
        while(current_S){ 
            if (current_S->hashValue == hash_S && current_S->key == lo_key_S) { // STALLS!
                ////// PROBE C JOIN CONDITION //////
                const int lo_key_C = lo_custkey[probeColTupleIdx];
                const uint32_t hash_C = hashInt32(lo_key_C);
                const size_t pos_C = hash_C & cachedView_C->htMask;
                current_C = reinterpret_cast<GrowingBufEntryScan*>(filterTagged(cachedView_C->ht[pos_C], hash_C)); // we have one view here (can have more in case of joins) // Global load (uncoalesced)
                while(current_C){ 
                    if (current_C->hashValue == hash_C && current_C->key == lo_key_C) {
                        ////// PROBE P JOIN CONDITION //////
                        const int lo_key_P = lo_partkey[probeColTupleIdx];
                        const uint32_t hash_P = hashInt32(lo_key_P);
                        const size_t pos_P = hash_P & cachedView_P->htMask;
                        GrowingBufEntryScan* current_P = reinterpret_cast<GrowingBufEntryScan*>(filterTagged(cachedView_P->ht[pos_P], hash_P));
                        while(current_P){
                            if (current_P->hashValue == hash_P && current_P->key == lo_key_P) {
                                ////// PROBE D JOIN CONDITION //////
                                const int lo_key_D = lo_orderdate[probeColTupleIdx];
                                const uint32_t hash_D = hashInt32(lo_key_D);
                                const size_t pos_D = hash_D & cachedView_D->htMask;
                                current_D = reinterpret_cast<GrowingBufEntryScan*>(filterTagged(cachedView_D->ht[pos_D], hash_D));
                                while(current_D){
                                    if(current_D->hashValue == hash_D && current_D->key == lo_key_D){
                                        foundMatch = true;
                                    }
                                    if(foundMatch){break;}
                                    current_D = current_D->next;
                                }
                            }
                            if(foundMatch){break;}
                            current_P = current_P->next;
                        } 
                    }
                    if(foundMatch){break;}
                    current_C = current_C->next;
                }
            }
            if(foundMatch){break;}
            current_S = current_S->next;
        }
        ////// [END] PROBE JOIN CONDITIONS //////
        }
        ////// INSERT/UPDATE PARTIAL AGGREGATE //////
        bool needInsert{false};
        int64_t hashGroupCols{-1};
        int scracthPadPos{-1};
        int outputPos{-1};
        GrowingBufEntryResHT* partialAggEntry;
        if(foundMatch){
            hashGroupCols = combineHashes(hashInt32(current_D->value), hashInt32(current_C->value));
            scracthPadPos = (hashGroupCols >> htShift) & scracthPadMask;
            outputPos = hashGroupCols & PreAggregationHashtableFragmentSMEM::outputMask;
            partialAggEntry = reinterpret_cast<GrowingBufEntryResHT*>(scracthPad[scracthPadPos]);
            if(!partialAggEntry){ 
                needInsert = true; // if no entry found (nullptr) at position
            } else {
                if(partialAggEntry->hashValue == hashGroupCols){ // Global load (stalls)
                    // Q4.1. returns select d_year,c_nation,sum(lo_revenue-lo_supplycost) as profit 
                    if(partialAggEntry->key[0] == current_D->value && partialAggEntry->key[1] == current_C->value){
                        needInsert = false; // if found entry, hash and key match
                    } else { 
                        needInsert = true; // if key doesn't match, collision -> insert
                    }
                } else { 
                    needInsert = true; // if hash doesn't match
                }
            }
        }
        uint32_t myIdx{-1};
        int64_t value{-1};
        // cg::coalesced_group sameOutputPosGroup = cg::labeled_partition(active, outputPos);
        if(foundMatch){
            value = lo_revenue[probeColTupleIdx] - lo_supplycost[probeColTupleIdx];
            if(needInsert){
                myIdx = atomicAdd(&myFrag->counters[outputPos], 1);
            } else {
                atomicAdd(reinterpret_cast<unsigned long long*>(&partialAggEntry->value), (long long)(value));
            }
        } 
        block.sync();
        for(int i = block.thread_rank(); i < PreAggregationHashtableFragmentSMEM::numOutputs; i+=block.size()){
            // With accumulated counters, we pick per-partition thread that exclusively requests memory
            myFrag->insertN(i); // thread-block sequence for an output is allocated
        }
        block.sync();

        if(foundMatch && needInsert){
            GrowingBufEntryResHT* myOffset = reinterpret_cast<GrowingBufEntryResHT*>(myFrag->writeOffsets[outputPos]); // get allocated sequence for the output pos
            myOffset[myIdx].hashValue = hashGroupCols;
            myOffset[myIdx].key[0] = current_D->value; // index into the allocated sequence
            myOffset[myIdx].key[1] = current_C->value;
            myOffset[myIdx].value = value;
            myOffset[myIdx].next=nullptr;
            atomicExch((unsigned long long*)&scracthPad[scracthPadPos], (unsigned long long)&myOffset[myIdx]);
        }
        // block.sync();
        // for(int i = block.thread_rank(); i < PreAggregationHashtableFragmentSMEM::numOutputs; i+=block.size()){
        //     // With accumulated counters, we pick per-partition thread that exclusively requests memory
        // }
        // block.sync();
        ////// [END] INSERT/UPDATE PARTIAL AGGREGATE //////
    }
    block.sync();

    if(!warpLane){
        int su = 0;
        for(int i = 0; i < PreAggregationHashtableFragment::numOutputs; i++){
            if(myFrag->outputs[i]){
                atomicAdd((unsigned long long*)&htSizes[i], (unsigned long long)myFrag->outputs[i]->getLen());
                su += myFrag->outputs[i]->getLen();
            }
            // printf("[buildPreAggr] outputs[%d] = %p\n", i, myFrag->outputs[i]);
        }
        memcpy(&globalOutputs[blockIdx.x * PreAggregationHashtableFragment::numOutputs], myFrag->outputs, sizeof(FlexibleBuffer *) * PreAggregationHashtableFragment::numOutputs);
    }
    ///////////////////////////////////////////
    // if(!threadIdx.x){myFrag->print(printEntryResHT);}  // only for <<<1,X>>> debug

    // if(!threadIdx.x){ // check that preAggrHT is initialized
    //     buildStats.print();
    // }
}  
*/
constexpr int64_t highestPowerOfTwo(int64_t n) { return n == 0 ? 0 : 1LL << (63 - __builtin_clzll(n));}
constexpr uint8_t powerOfTwo(int64_t n, int power = 0) {return (n == 1) ? power : powerOfTwo(n / 2, power + 1);}
constexpr int64_t SMEM_SIZE{36 * KiB};
static constexpr int64_t freeSMEM{SMEM_SIZE - (sizeof(PreAggregationHashtableFragmentSMEM) + 4*sizeof(HashIndexedView))}; 
static constexpr uint8_t scracthPadShift{powerOfTwo(highestPowerOfTwo(freeSMEM)/sizeof(PreAggregationHashtableFragmentSMEM::Entry*))};
static constexpr uint64_t scracthPadSize{1 << scracthPadShift};
static constexpr uint64_t scracthPadMask{scracthPadSize-1};
__global__ void buildPreAggregationHashtableFragmentsAdvanced(
        int* lo_orderdate, int* lo_partkey, int* lo_custkey, int* lo_suppkey, int* lo_revenue, int* lo_supplycost, int lo_len,
        HashIndexedView* sView, HashIndexedView* cView, HashIndexedView* pView, HashIndexedView* dView, 
        PreAggregationHashtableFragmentSMEM* fragments) 
    {
    //  1024 threads per block, the maximum registers per thread is 64
    const int warpCount = (blockDim.x + (WARP_SIZE-1)) / WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;
    const int warpLane = threadIdx.x % WARP_SIZE;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    const int globalWarpID = globalTid / WARP_SIZE;

    __shared__ char smem[
        scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*) 
        + 4*sizeof(HashIndexedView) 
        + sizeof(PreAggregationHashtableFragmentSMEM)];
    PreAggregationHashtableFragmentSMEM::Entry** scracthPad = reinterpret_cast<PreAggregationHashtableFragmentSMEM::Entry**>(smem);
    HashIndexedView* cachedView_S = reinterpret_cast<HashIndexedView*>(smem + scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*));
    HashIndexedView* cachedView_P = reinterpret_cast<HashIndexedView*>(smem + scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*) + sizeof(HashIndexedView));
    HashIndexedView* cachedView_D = reinterpret_cast<HashIndexedView*>(smem + scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*) + 2*sizeof(HashIndexedView));
    HashIndexedView* cachedView_C = reinterpret_cast<HashIndexedView*>(smem + scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*) + 3*sizeof(HashIndexedView));
    PreAggregationHashtableFragmentSMEM* myFrag = reinterpret_cast<PreAggregationHashtableFragmentSMEM*>(smem + scracthPadSize * sizeof(PreAggregationHashtableFragmentSMEM::Entry*) + 4*sizeof(HashIndexedView));

    for(int i = threadIdx.x; i < scracthPadSize; i+=blockDim.x){
        scracthPad[i] = nullptr;
    }
    if(threadIdx.x == 0){
        *cachedView_S = *sView;
        *cachedView_P = *pView;
        *cachedView_D = *dView;
        *cachedView_C = *cView;
        new(myFrag) PreAggregationHashtableFragmentSMEM(TYPE_SIZE_RES_HT);
    }
    __syncthreads();
    // iterate over probe cols
    int probeColIdxStart = globalTid;
    int probeColIdxStep = numThreadsTotal;
    int roundedSize = ((lo_len + 31) / 32) * 32; 
    for(int probeColTupleIdx = probeColIdxStart; probeColTupleIdx < roundedSize; probeColTupleIdx+=probeColIdxStep){
        const bool remainInLoop{probeColTupleIdx < lo_len};
        bool foundMatch{false}; // PROBING
        GrowingBufEntryScan* current_C{nullptr}; // value cols
        GrowingBufEntryScan* current_D{nullptr}; 
        if(remainInLoop){
        ////// PROBE S JOIN CONDITION //////
        const int lo_key_S = lo_suppkey[probeColTupleIdx];
        const uint64_t hash_S = hashInt32ToInt64(lo_key_S);
        const size_t pos_S = hash_S & cachedView_S->htMask;
        GrowingBufEntryScan* current_S = reinterpret_cast<GrowingBufEntryScan*>(filterTagged(cachedView_S->ht[pos_S], hash_S)); // we have one view here (can have more in case of joins) // Global load (uncoalesced)
        while(current_S){ 
            if (current_S->hashValue == hash_S && current_S->key == lo_key_S) { // STALLS!
                ////// PROBE C JOIN CONDITION //////
                const int lo_key_C = lo_custkey[probeColTupleIdx];
                const uint64_t hash_C = hashInt32ToInt64(lo_key_C);
                const size_t pos_C = hash_C & cachedView_C->htMask;
                current_C = reinterpret_cast<GrowingBufEntryScan*>(filterTagged(cachedView_C->ht[pos_C], hash_C)); // we have one view here (can have more in case of joins) // Global load (uncoalesced)
                while(current_C){ 
                    if (current_C->hashValue == hash_C && current_C->key == lo_key_C) {
                        ////// PROBE P JOIN CONDITION //////
                        const int lo_key_P = lo_partkey[probeColTupleIdx];
                        const uint64_t hash_P = hashInt32ToInt64(lo_key_P);
                        const size_t pos_P = hash_P & cachedView_P->htMask;
                        GrowingBufEntryScan* current_P = reinterpret_cast<GrowingBufEntryScan*>(filterTagged(cachedView_P->ht[pos_P], hash_P));
                        while(current_P){
                            if (current_P->hashValue == hash_P && current_P->key == lo_key_P) {
                                ////// PROBE D JOIN CONDITION //////
                                const int lo_key_D = lo_orderdate[probeColTupleIdx];
                                const uint64_t hash_D = hashInt32ToInt64(lo_key_D);
                                const size_t pos_D = hash_D & cachedView_D->htMask;
                                current_D = reinterpret_cast<GrowingBufEntryScan*>(filterTagged(cachedView_D->ht[pos_D], hash_D));
                                while(current_D){
                                    if(current_D->hashValue == hash_D && current_D->key == lo_key_D){
                                        foundMatch = true;
                                    }
                                    if(foundMatch){break;}
                                    current_D = filterTagged(current_D->next, hash_D);
                                }
                            }
                            if(foundMatch){break;}
                            current_P = filterTagged(current_P->next, hash_P);
                        } 
                    }
                    if(foundMatch){break;}
                    current_C = filterTagged(current_C->next, hash_C);
                }
            }
            if(foundMatch){break;}
            current_S = filterTagged(current_S->next, hash_S);
        }
        ////// [END] PROBE JOIN CONDITIONS //////
        }

        ////// INSERT/UPDATE PARTIAL AGGREGATE //////
        bool needInsert{false};
        int64_t hashGroupCols{-1};
        int scracthPadPos{-1};
        GrowingBufEntryResHT* partialAggEntry = nullptr;
        if(foundMatch){
            hashGroupCols = combineHashes(hashInt32ToInt64(current_D->value), hashInt32ToInt64(current_C->value));
            scracthPadPos = (hashGroupCols >> scracthPadShift) & scracthPadMask;
            partialAggEntry = reinterpret_cast<GrowingBufEntryResHT*>(scracthPad[scracthPadPos]);
            if(!partialAggEntry){ 
                needInsert = true; // if no entry found (nullptr) at position
            } else {
                if(partialAggEntry->hashValue == hashGroupCols){ // Global load (stalls)
                    // Q4.1. returns select d_year,c_nation,sum(lo_revenue-lo_supplycost) as profit 
                    if(partialAggEntry->key[0] == current_D->value && partialAggEntry->key[1] == current_C->value){
                        needInsert = false; // if found entry, hash and key match
                    } else { 
                        needInsert = true; // if key doesn't match, collision -> insert
                    }
                } else { 
                    needInsert = true; // if hash doesn't match
                }
            }
        }
        int mask = __ballot_sync(0xFFFFFFFF, foundMatch && needInsert);
        if(foundMatch){
            int64_t value = lo_revenue[probeColTupleIdx] - lo_supplycost[probeColTupleIdx];
            if(needInsert){
                GrowingBufEntryResHT* myEntry = reinterpret_cast<GrowingBufEntryResHT*>(myFrag->insertWarpOpportunistic(hashGroupCols, mask));
                myEntry->hashValue = hashGroupCols;
                myEntry->key[0] = current_D->value;
                myEntry->key[1] = current_C->value;
                myEntry->value = value;
                myEntry->next=nullptr;
                atomicExch_block(reinterpret_cast<unsigned long long*>(&scracthPad[scracthPadPos]), (unsigned long long)myEntry);
            } else {
                atomicAdd(reinterpret_cast<unsigned long long*>(&partialAggEntry->value), (long long)(value));
                
                // Complex values may be unable to use atomics -> lock
                // GrowingBufEntryResHT* next;
                // do{
                //     next = (GrowingBufEntryResHT*)atomicExch((unsigned long long*)&partialAggEntry->next, 1ull);
                // } while((unsigned long long)next == 1ull);
                // partialAggEntry->value += value;
                // atomicExch((unsigned long long*)&partialAggEntry->next, (unsigned long long)next);
            }
        } 
        ////// [END] INSERT/UPDATE PARTIAL AGGREGATE //////
    }
    __syncthreads();
    for(uint32_t i = threadIdx.x; i < sizeof(PreAggregationHashtableFragmentSMEM); i+=blockDim.x){ // cooperate on copy
        char* myFragAsChar = reinterpret_cast<char*>(myFrag);
        char* globalFragAsChar = reinterpret_cast<char*>(&fragments[blockIdx.x]);
        globalFragAsChar[i] = myFragAsChar[i];
    }
    ///////////////////////////////////////////
}  

__global__ void printPreAggregationHashtable(PreAggregationHashtable* ht, bool printEmpty=false) {
    printf("---------------------PreAggregationHashtable [%p]-------------------------\n", ht);
    int resCnt{0};
    for(int p = 0; p < PreAggregationHashtableFragment::numOutputs; p++){
        for(int i = 0; i < ht->ht[p].hashMask+1; i++){
            GrowingBufEntryResHT* curr = reinterpret_cast<GrowingBufEntryResHT*>(ht->ht[p].ht[i]);
            if(!printEmpty && !curr){continue;}
            printf("[PARTITION %d, htEntryIdx=%d]", p, i);
            while(curr){
                printf(", {ptr=%p, next=%p, KEY1: %d, KEY2: %d, AGG: %lld}", curr, curr->next, curr->key[0], curr->key[1], curr->value);
                curr = curr->next;
                resCnt++;
            }
            printf("\n");
        }
    }
    printf("Res count: %d\n", resCnt);
    printf("------------------[END] PreAggregationHashtable [%p]----------------------\n", ht);
}


struct ResHTEntryContent{
    bool nullFlag;
    int32_t key[2]; // d_year,c_nation ... group by d_year,c_nation
    int64_t value; // sum(lo_revenue-lo_supplycost)
};
__device__ bool eqInt(uint8_t* lhs, uint8_t* rhs){
    auto* lhsC = reinterpret_cast<GrowingBufEntryResHT*>(lhs);
    auto* rhsC = reinterpret_cast<GrowingBufEntryResHT*>(rhs);
    return lhsC->key[0] == rhsC->key[0] && lhsC->key[1] == rhsC->key[1];
}
__device__ void combineInt(uint8_t* lhs, uint8_t* rhs){
    auto* lhsC = reinterpret_cast<GrowingBufEntryResHT*>(lhs);
    auto* rhsC = reinterpret_cast<GrowingBufEntryResHT*>(rhs);
    lhsC->value += rhsC->value;
}
__global__ void INITPreAggregationHashtableFragmentsSingleThread(PreAggregationHashtable* preAggrHT, PreAggregationHashtable::PartitionHt* preAllocatedPartitions){
    if(blockDim.x * blockIdx.x + threadIdx.x == 0){
        new(preAggrHT) PreAggregationHashtable(preAllocatedPartitions);
    }
}
__global__ void mergePreAggregationHashtableFragments(
        PreAggregationHashtable* preAggrHT, 
        PreAggregationHashtable::PartitionHt* preAllocatedPartitions, 
        PreAggregationHashtableFragmentSMEM* fragments, 
        size_t numFrags) 
    {
    const int warpCount = (blockDim.x + (WARP_SIZE-1)) / WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;
    const int warpLane = threadIdx.x % WARP_SIZE;
    const int numThreadsTotal = blockDim.x * gridDim.x;
    const int globalTid = blockDim.x * blockIdx.x + threadIdx.x;

    int cntr{0};
    /*
        - Partitions: have hts that are mutually exclusive in terms of sync -> partition-to-SM
        - Fragments: 
    */
    int partitionId = blockIdx.x % 64;
    int partitionWorkerId = blockIdx.x / 64;

    int blocks_per_partition = gridDim.x / 64;
    int extra_blocks = gridDim.x % 64;
    int stride = blocks_per_partition + (partitionId <= extra_blocks);

    PreAggregationHashtable::Entry** ht = preAggrHT->ht[partitionId].ht;
    const size_t htMask = preAggrHT->ht[partitionId].hashMask;
    // __syncthreads();
    // printf("[MergePreAggr] numFrags=%lu\n", numFrags);
    for(int fragmentId = 0; fragmentId < numFrags; fragmentId++){ 
        FlexibleBuffer* fragmentPartitionBuffer = fragments[fragmentId].getPartitionPtr(partitionId);
        // printf("[MergePreAggr][fragmentId=%d] fragmentPartitionBuffer=%p\n",fragmentId, fragmentPartitionBuffer);
        if(!fragmentPartitionBuffer->getTypeSize()){continue;} // many stalls, long scoreboard
        const int buffersCnt{fragmentPartitionBuffer->getBuffers().size()};
        for(int bufferIdx = partitionWorkerId; bufferIdx < buffersCnt; bufferIdx+=stride){
            Buffer* buf = &fragmentPartitionBuffer->getBuffers().payLoad[bufferIdx];
            const int elemsCnt{buf->numElements};
            for (int elementIdx = threadIdx.x; elementIdx < buf->numElements; elementIdx+=blockDim.x) {
                PreAggregationHashtableFragment::Entry* curr = reinterpret_cast<PreAggregationHashtableFragment::Entry*>(&buf->ptr[elementIdx * TYPE_SIZE_RES_HT]); // Global load
                const size_t pos = curr->hashValue >> PreAggregationHashtableFragment::htShift & htMask; // Global load
                
                // printf("[Partition %d][POS %lu] MERGING hash=%llu, key1=%d, key2=%d\n", partitionId, pos, p->hashValue, p->key[0], p->key[1]);
                // PreAggregationHashtable::Entry* currCandidate = untag(ht[pos]);
                PreAggregationHashtable::Entry* currCandidate;
                do{
                    currCandidate = reinterpret_cast<PreAggregationHashtable::Entry*>(atomicExch((unsigned long long*)&ht[pos], 1ull)); // global write, long scoreboards
                }
                while((unsigned long long)currCandidate == 1ull);

                bool merged = false;
                auto* currPtr = reinterpret_cast<GrowingBufEntryResHT*>(curr);
                while (currCandidate) {
                    auto* candidatePtr = reinterpret_cast<GrowingBufEntryResHT*>(currCandidate);
                    if (currCandidate->hashValue == curr->hashValue && eqInt((uint8_t*)candidatePtr, (uint8_t*)currPtr)) { // Global loads, stalls, bad L2
                        combineInt((uint8_t*)candidatePtr, (uint8_t*)currPtr); // bad L2
                        merged = true;
                        break;
                    }
                    currCandidate = currCandidate->next;
                }
                if (!merged) {
                    PreAggregationHashtable::Entry* previousPtr = currCandidate;
                    currCandidate = tag(curr, previousPtr, curr->hashValue);
                    currCandidate = curr;
                    curr->next = untag(previousPtr);
                }
                atomicExch((unsigned long long*)&ht[pos], (unsigned long long)currCandidate);
                // if(atomicCAS((unsigned long long*)&ht[pos], 1ull, (unsigned long long)currCandidate) != 1ull){
                    // printf("Trouble\n");
                // }
            }
        }

    }
    // acquire_lock(&preAggrHT->mutex);
    // // Append buffers that back partition's pointers (no invalidation, because buffer itself is not reallocated)
    // preAggrHT->buffer.merge(localBuffer); 
    // release_lock(&preAggrHT->mutex);
}

__global__ void freeKernel(GrowingBuffer* finalBuffer) {
    finalBuffer->~GrowingBuffer();
}

__global__ void freeFragments(PreAggregationHashtableFragment* partitions, int numPartitions) {
    for(int i = 0; i < numPartitions; i++){
        partitions[i].~PreAggregationHashtableFragment();
    }
}

float q41(int* lo_orderdate, int* lo_custkey, int* lo_partkey, int* lo_suppkey, int* lo_revenue, int* lo_supplycost, int lo_len,
    int *d_datekey, int* d_year, int d_len,
    int *p_partkey, int* p_mfgr, int p_len,
    int *s_suppkey, int* s_region, int s_len,
    int *c_custkey, int* c_region, int* c_nation, int c_len){
        std::cout << "** BUILDING HASH VIEWS **" << std::endl;
        ViewResult sView = buildView<TABLE::S>(s_region, s_suppkey, nullptr, s_len);
        ViewResult cView = buildView<TABLE::C>(c_region, c_custkey, c_nation, c_len);
        ViewResult pView = buildView<TABLE::P>(p_mfgr, p_partkey, nullptr, p_len);
        ViewResult dView = buildView<TABLE::D>(nullptr, d_datekey, d_year, d_len);
        std::cout << "** BUILT HASH VIEWS **" << std::endl;
        std::cout << "** BUILDING PREAGGREGATION FRAGMENTS **" << std::endl;
        const size_t numFragments = 30;
        const size_t numThreadsInBlockPreAggr = 1024;
        PreAggregationHashtableFragmentSMEM* fragments_d;
        PreAggregationHashtableFragmentSMEM* fragments_h;
        CHECK_CUDA_ERROR(hipHostMalloc(&fragments_h, numFragments * sizeof(PreAggregationHashtableFragmentSMEM)));
        CHECK_CUDA_ERROR(hipMalloc(&fragments_d, numFragments * sizeof(PreAggregationHashtableFragmentSMEM)));
        // std::cout << "[buildPreAggregationHashtableFragments] Launch config: <<<" <<numBlocks << ","<<numThreadsInBlockPreAggr <<  ">>>\n";
        buildPreAggregationHashtableFragmentsAdvanced<<<numFragments, numThreadsInBlockPreAggr>>>(
            lo_orderdate, lo_partkey, lo_custkey, lo_suppkey, lo_revenue, lo_supplycost, lo_len,
            sView.d_hash_view, cView.d_hash_view, pView.d_hash_view, dView.d_hash_view, 
            fragments_d
        );
        hipDeviceSynchronize();
        CHECK_CUDA_ERROR(hipGetLastError());

        std::cout << "** BUILT PREAGGREGATION FRAGMENTS **" << std::endl;
        std::cout << "** MERGING PREAGGREGATION FRAGMENTS **" << std::endl;

        CHECK_CUDA_ERROR(hipMemcpy(fragments_h, fragments_d, numFragments * sizeof(PreAggregationHashtableFragmentSMEM), hipMemcpyDeviceToHost)); // get sizes back
        PreAggregationHashtable::PartitionHt* d_preAllocatedPartitions;
        PreAggregationHashtable::PartitionHt* h_preAllocatedPartitions;
        CHECK_CUDA_ERROR(hipMalloc(&d_preAllocatedPartitions, sizeof(PreAggregationHashtable::PartitionHt) * 64));
        CHECK_CUDA_ERROR(hipHostMalloc(&h_preAllocatedPartitions, sizeof(PreAggregationHashtable::PartitionHt) * 64));

        for(int partitionID = 0; partitionID < PreAggregationHashtableFragmentSMEM::numPartitions; partitionID++){
            uint64_t partitionSize = 0;
            for(int fragId = 0; fragId < numFragments; fragId++){
                partitionSize += fragments_h[fragId].getPartitionPtr(partitionID)->getLen();
            }
            auto [htAllocSize, htMask] = getHtSizeMask(partitionSize, sizeof(PreAggregationHashtableFragment::Entry*));
            h_preAllocatedPartitions[partitionID].hashMask = htMask;
            CHECK_CUDA_ERROR(hipMalloc(&h_preAllocatedPartitions[partitionID].ht, htAllocSize));
            CHECK_CUDA_ERROR(hipMemset(h_preAllocatedPartitions[partitionID].ht, 0, htAllocSize));
        }
        // std::cout << "[Merge HT FRAGMENTS] Total size = " << totalSum << ", num fragments " << numFragments << "\n";
        CHECK_CUDA_ERROR(hipMemcpy(d_preAllocatedPartitions, h_preAllocatedPartitions, sizeof(PreAggregationHashtable::PartitionHt) * 64, hipMemcpyHostToDevice));
        
        PreAggregationHashtable* h_result_preAggrHT;
        PreAggregationHashtable* d_result_preAggrHT;
        CHECK_CUDA_ERROR(hipHostMalloc(&h_result_preAggrHT, sizeof(PreAggregationHashtable)));
        CHECK_CUDA_ERROR(hipMalloc(&d_result_preAggrHT, sizeof(PreAggregationHashtable)));

        INITPreAggregationHashtableFragmentsSingleThread<<<1,1>>>(d_result_preAggrHT, d_preAllocatedPartitions);
        mergePreAggregationHashtableFragments<<<64,512>>>(d_result_preAggrHT, d_preAllocatedPartitions, fragments_d, numFragments);
        printPreAggregationHashtable<<<1,1>>>(d_result_preAggrHT, false);
        hipDeviceSynchronize();
        CHECK_CUDA_ERROR(hipGetLastError());


        std::cout << "** MERGED PREAGGREGATION FRAGMENTS **" << std::endl;

        // Free heap allocations:
        freeKernel<<<1,1>>>(sView.d_filter_scan);
        freeKernel<<<1,1>>>(cView.d_filter_scan);
        freeKernel<<<1,1>>>(pView.d_filter_scan);
        freeKernel<<<1,1>>>(dView.d_filter_scan);
        CHECK_CUDA_ERROR(hipFree(sView.h_hash_view->ht));
        CHECK_CUDA_ERROR(hipFree(cView.h_hash_view->ht));
        CHECK_CUDA_ERROR(hipFree(pView.h_hash_view->ht));
        CHECK_CUDA_ERROR(hipFree(dView.h_hash_view->ht));

        hipDeviceSynchronize();
        CHECK_CUDA_ERROR(hipGetLastError());

        CHECK_CUDA_ERROR(hipHostFree(h_result_preAggrHT));
        CHECK_CUDA_ERROR(hipFree(d_result_preAggrHT));
        
        CHECK_CUDA_ERROR(hipHostFree(h_preAllocatedPartitions));
        CHECK_CUDA_ERROR(hipFree(d_preAllocatedPartitions));

        CHECK_CUDA_ERROR(hipFree(fragments_d));
        CHECK_CUDA_ERROR(hipHostFree(fragments_h));

        for(int outputId = 0; outputId < PreAggregationHashtableFragment::numOutputs; outputId++){
            CHECK_CUDA_ERROR(hipFree(h_preAllocatedPartitions[outputId].ht));
        }

        CHECK_CUDA_ERROR(hipFree(sView.d_hash_view));
        CHECK_CUDA_ERROR(hipFree(sView.d_filter_scan));
        CHECK_CUDA_ERROR(hipHostFree(sView.h_filter_scan));
        CHECK_CUDA_ERROR(hipHostFree(sView.h_hash_view));

        CHECK_CUDA_ERROR(hipFree(cView.d_hash_view));
        CHECK_CUDA_ERROR(hipFree(cView.d_filter_scan));
        CHECK_CUDA_ERROR(hipHostFree(cView.h_filter_scan));
        CHECK_CUDA_ERROR(hipHostFree(cView.h_hash_view));

        CHECK_CUDA_ERROR(hipFree(pView.d_hash_view));
        CHECK_CUDA_ERROR(hipFree(pView.d_filter_scan));
        CHECK_CUDA_ERROR(hipHostFree(pView.h_filter_scan));
        CHECK_CUDA_ERROR(hipHostFree(pView.h_hash_view));

        CHECK_CUDA_ERROR(hipFree(dView.d_hash_view));
        CHECK_CUDA_ERROR(hipFree(dView.d_filter_scan));
        CHECK_CUDA_ERROR(hipHostFree(dView.h_filter_scan));
        CHECK_CUDA_ERROR(hipHostFree(dView.h_hash_view));

        return 1.1;
    }


//////////////////////////////////////////////// QUERY 4.1 ////////////////////////////////////////////////


int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " dataSetPath SF numBlocks \n";
        return 1;
    }
    const std::string dataSetPath = argv[1];
    sf = std::atoi(argv[2]);
    int numBlocks = std::atoi(argv[3]);
    initialize(sf);

    std::cout << "** LOADING DATA  CPU **" << std::endl;

    int *h_lo_orderdate = loadColumn<int>(dataSetPath,h_DATA_DIR,"lo_orderdate", h_LO_LEN);
    int *h_lo_suppkey = loadColumn<int>(dataSetPath,h_DATA_DIR,"lo_suppkey", h_LO_LEN);
    int *h_lo_custkey = loadColumn<int>(dataSetPath,h_DATA_DIR,"lo_custkey", h_LO_LEN);
    int *h_lo_partkey = loadColumn<int>(dataSetPath,h_DATA_DIR,"lo_partkey", h_LO_LEN);
    int *h_lo_revenue = loadColumn<int>(dataSetPath,h_DATA_DIR,"lo_revenue", h_LO_LEN);
    int *h_lo_supplycost = loadColumn<int>(dataSetPath,h_DATA_DIR,"lo_supplycost", h_LO_LEN);

    int *h_d_datekey = loadColumn<int>(dataSetPath,h_DATA_DIR,"d_datekey", h_D_LEN);
    int *h_d_year = loadColumn<int>(dataSetPath,h_DATA_DIR,"d_year", h_D_LEN);
    int *h_d_yearmonthnum = loadColumn<int>(dataSetPath,h_DATA_DIR,"d_yearmonthnum", h_D_LEN);

    int *h_s_suppkey = loadColumn<int>(dataSetPath,h_DATA_DIR,"s_suppkey", h_S_LEN);
    int *h_s_region = loadColumn<int>(dataSetPath,h_DATA_DIR,"s_region", h_S_LEN);

    int *h_p_partkey = loadColumn<int>(dataSetPath,h_DATA_DIR,"p_partkey", h_P_LEN);
    int *h_p_mfgr = loadColumn<int>(dataSetPath,h_DATA_DIR,"p_mfgr", h_P_LEN);

    int *h_c_custkey = loadColumn<int>(dataSetPath,h_DATA_DIR,"c_custkey", h_C_LEN);
    int *h_c_region = loadColumn<int>(dataSetPath,h_DATA_DIR,"c_region", h_C_LEN);
    int *h_c_nation = loadColumn<int>(dataSetPath,h_DATA_DIR,"c_nation", h_C_LEN);

    std::cout << "** LOADED DATA **" << std::endl;

    int *d_lo_orderdate = loadToGPU<int>(h_lo_orderdate, h_LO_LEN);
    int *d_lo_custkey = loadToGPU<int>(h_lo_custkey, h_LO_LEN);
    int *d_lo_suppkey = loadToGPU<int>(h_lo_suppkey, h_LO_LEN);
    int *d_lo_partkey = loadToGPU<int>(h_lo_partkey, h_LO_LEN);
    int *d_lo_revenue = loadToGPU<int>(h_lo_revenue, h_LO_LEN);
    int *d_lo_supplycost = loadToGPU<int>(h_lo_supplycost, h_LO_LEN);

    int *d_d_datekey = loadToGPU<int>(h_d_datekey, h_D_LEN);
    int *d_d_year = loadToGPU<int>(h_d_year, h_D_LEN);

    int *d_p_partkey = loadToGPU<int>(h_p_partkey, h_P_LEN);
    int *d_p_mfgr = loadToGPU<int>(h_p_mfgr, h_P_LEN);

    int *d_s_suppkey = loadToGPU<int>(h_s_suppkey, h_S_LEN);
    int *d_s_region = loadToGPU<int>(h_s_region, h_S_LEN);

    int *d_c_custkey = loadToGPU<int>(h_c_custkey, h_C_LEN);
    int *d_c_region = loadToGPU<int>(h_c_region, h_C_LEN);
    int *d_c_nation = loadToGPU<int>(h_c_nation, h_C_LEN);

    std::cout << "** LOADED DATA TO GPU **" << std::endl;

    #ifdef GALLATIN_ENABLED
    gallatin::allocators::init_global_allocator(HEAP_SIZE, 10, false);
    #else
    hipDeviceSetLimit(hipLimitMallocHeapSize, HEAP_SIZE);
    #endif
    for (int t = 0; t < NUM_RUNS; t++) {
        float time_query = q41(  
            d_lo_orderdate, d_lo_custkey, d_lo_partkey, d_lo_suppkey, d_lo_revenue, d_lo_supplycost, h_LO_LEN,
            d_d_datekey, d_d_year, h_D_LEN,
            d_p_partkey, d_p_mfgr, h_P_LEN,
            d_s_suppkey, d_s_region, h_S_LEN,
            d_c_custkey, d_c_region, d_c_nation, h_C_LEN);
        std::cout << "Time: " << time_query << "\n";
    }
    return 0;
}